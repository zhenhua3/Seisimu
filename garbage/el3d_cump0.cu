#include "hip/hip_runtime.h"
#include<stdio.h>
#include"fd_body_bound.c"

extern "C"{
  __global__ void kernel_v(double *dev_vx, double *dev_rho,
  double *dev_txx, double *dev_txy, double *dev_txz,
  double dx, double dy, double dz,
  int nx_vx, int ny_vx, int nz_vx,
  int nx_tpp, int ny_tpp, int nz_tpp,
  int nx_txy, int ny_txy, int nz_txy,
  int nx_txz, int ny_txz, int nz_txz,
  double *dev_fdc, int chunk)
  {
    int tX = threadIdx.x + blockIdx.x*blockDim.x;
    int tY = threadIdx.y + blockIdx.y*blockDim.y;
    int tZ = threadIdx.z + blockIdx.z*blockDim.z;

    // vx
    int tid_vx = tZ + tX*nz_vx + tY*nx_vx*nz_vx;

    int tid_txx_0 = tZ + (tX+0)*nz_tpp + tY*nx_txx*nz_tpp;
    int tid_txx_1 = tZ + (tX+1)*nz_tpp + tY*nx_tpp*nz_tpp;
    int tid_txx_2 = tZ + (tX+2)*nz_tpp + tY*nx_tpp*nz_tpp;
    int tid_txx_3 = tZ + (tX+3)*nz_tpp + tY*nx_tpp*nz_tpp;

    int tid_txy_0 = tZ + (tX+2)*nz_txy + (tY+0)*nx_txy*nz_txy;
    int tid_txy_1 = tZ + (tX+2)*nz_txy + (tY+1)*nx_txy*nz_txy;
    int tid_txy_2 = tZ + (tX+2)*nz_txy + (tY+2)*nx_txy*nz_txy;
    int tid_txy_3 = tZ + (tX+2)*nz_txy + (tY+3)*nx_txy*nz_txy;

    int tid_txz_0 = (tZ+0) + (tX+2)*nz_txz + tY*nx_txz*nz_txz;
    int tid_txz_1 = (tZ+1) + (tX+2)*nz_txz + tY*nx_txz*nz_txz;
    int tid_txz_2 = (tZ+2) + (tX+2)*nz_txz + tY*nx_txz*nz_txz;
    int tid_txz_3 = (tZ+3) + (tX+2)*nz_txz + tY*nx_txz*nz_txz;

    int tid_rho_vx_0 = tZ + (tX+0)*nz_tpp + tY*nx_txx*nz_tpp;
    int tid_rho_vx_1 = tZ + (tX+1)*nz_tpp + tY*nx_txx*nz_tpp;

    double tmp_vx;
    if(tX < nx_vx && tY < chunk && tZ < nz_vx){
      tmp_vx = (*(dev_txx+tid_txx_0)* *(dev_fdc)
      + *(dev_txx+tid_txx_1)* *(dev_fdc+1)
      + *(dev_txx+tid_txx_2)* *(dev_fdc+2)
      + *(dev_txx+tid_txx_3)* *(dev_fdc+3)) / dx;

      tmp_vx = tmp_vx + (*(dev_txy+tid_txy_0)* *(dev_fdc)
      + *(dev_txy+tid_txy_1)* *(dev_fdc+1)
      + *(dev_txy+tid_txy_2)* *(dev_fdc+2)
      + *(dev_txy+tid_txy_3)* *(dev_fdc+3)) / dy;

      tmp_vx = tmp_vx + (*(dev_txz+tid_txz_0)* *(dev_fdc)
      + *(dev_txz+tid_txz_1)* *(dev_fdc+1)
      + *(dev_txz+tid_txz_2)* *(dev_fdc+2)
      + *(dev_txz+tid_txz_3)* *(dev_fdc+3)) / dz;

      tmp_rho_vx = 2/(*(dev_rho + tid_rho_vx_0)
      + *(dev_rho + tid_rho_vx_1));

      *(dev_vx+tid_vx) = *(dev_vx+tid_vx) + tmp_rho_vx * tmp_vx;
    }

    // vy
    int tid_vy = tZ + tX*nz_vy + tY*nx_vy*nz_vy;
    int tid_txy_0 = tZ + (tX+0)*nz_txy + (tY+2)*nx_txy*nz_txy;
    int tid_txy_1 = tZ + (tX+1)*nz_txy + (tY+2)*nx_txy*nz_txy;
    int tid_txy_2 = tZ + (tX+2)*nz_txy + (tY+2)*nx_txy*nz_txy;
    int tid_txy_3 = tZ + (tX+3)*nz_txy + (tY+2)*nx_txy*nz_txy;

    int tid_tyy_0 = tZ + tX*nz_tyy + (tY+0)*nx_tyy*nz_tyy;
    int tid_tyy_1 = tZ + tX*nz_tyy + (tY+1)*nx_tyy*nz_tyy;
    int tid_tyy_2 = tZ + tX*nz_tyy + (tY+2)*nx_tyy*nz_tyy;
    int tid_tyy_3 = tZ + tX*nz_tyy + (tY+3)*nx_tyy*nz_tyy;

    int tid_tyz_0 = (tZ+0) + tX*nz_tyz + (tY+2)*nx_tyz*nz_tyz;
    int tid_tyz_1 = (tZ+1) + tX*nz_tyz + (tY+2)*nx_tyz*nz_tyz;
    int tid_tyz_2 = (tZ+2) + tX*nz_tyz + (tY+2)*nx_tyz*nz_tyz;
    int tid_tyz_3 = (tZ+3) + tX*nz_tyz + (tY+2)*nx_tyz*nz_tyz;

    int tid_rho_vy_0 = tZ + tX*nz_tyy + (tY+0)*nx_tyy*nz_tyy;
    int tid_rho_vy_1 = tZ + tX*nz_tyy + (tY+1)*nx_tyy*nz_tyy;

    double tmp_vy;
    if(tX < nx_vy && tY < chunk && tZ < nz_vy){
      tmp_vy = (*(dev_txy+tid_txy_0)* *(dev_fdc)
      + *(dev_txy+tid_txy_1)* *(dev_fdc+1)
      + *(dev_txy+tid_txy_2)* *(dev_fdc+2)
      + *(dev_txy+tid_txy_3)* *(dev_fdc+3)) / dx;

      tmp_vy = tmp_vy + (*(dev_tyy+tid_tyy_0)* *(dev_fdc)
      + *(dev_tyy+tid_tyy_1)* *(dev_fdc+1)
      + *(dev_tyy+tid_tyy_2)* *(dev_fdc+2)
      + *(dev_tyy+tid_tyy_3)* *(dev_fdc+3)) / dy;

      tmp_vy = tmp_vy + (*(dev_tyz+tid_tyz_0)* *(dev_fdc)
      + *(dev_tyz+tid_tyz_1)* *(dev_fdc+1)
      + *(dev_tyz+tid_tyz_2)* *(dev_fdc+2)
      + *(dev_tyz+tid_tyz_3)* *(dev_fdc+3)) / dz;

      tmp_rho_vy = 2/(*(dev_rho + tid_rho_vy_0)
      + *(dev_rho + tid_rho_vy_1));

      *(dev_vy+tid_vy) = *(dev_vy+tid_vy) + tmp_rho_vy * tmp_vy;
    }

    // vz
    int tid_vz = tZ + tX*nz_vz + tY*nx_vz*nz_vz;
    int tid_txz_0 = tZ+2 + (tX+0)*nz_txz + tY*nx_txz*nz_txz;
    int tid_txz_1 = tZ+2 + (tX+1)*nz_txz + tY*nx_txz*nz_txz;
    int tid_txz_2 = tZ+2 + (tX+2)*nz_txz + tY*nx_txz*nz_txz;
    int tid_txz_3 = tZ+2 + (tX+3)*nz_txz + tY*nx_txz*nz_txz;

    int tid_tyz_0 = tZ+2 + tX*nz_tyz + (tY+0)*nx_tyz*nz_tyz;
    int tid_tyz_1 = tZ+2 + tX*nz_tyz + (tY+1)*nx_tyz*nz_tyz;
    int tid_tyz_2 = tZ+2 + tX*nz_tyz + (tY+2)*nx_tyz*nz_tyz;
    int tid_tyz_3 = tZ+2 + tX*nz_tyz + (tY+3)*nx_tyz*nz_tyz;

    int tid_tzz_0 = (tZ+0) + tX*nz_tzz + tY*nx_tzz*nz_tzz;
    int tid_tzz_1 = (tZ+1) + tX*nz_tzz + tY*nx_tzz*nz_tzz;
    int tid_tzz_2 = (tZ+2) + tX*nz_tzz + tY*nx_tzz*nz_tzz;
    int tid_tzz_3 = (tZ+3) + tX*nz_tzz + tY*nx_tzz*nz_tzz;

    int tid_rho_vz_0 = (tZ+0) + tX*nz_tzz + tY*nx_tzz*nz_tzz;
    int tid_rho_vz_1 = (tZ+1) + tX*nz_tzz + tY*nx_tzz*nz_tzz;

    double tmp_vz;
    if(tX < nx_vz && tY < chunk && tZ < nz_vz){
      tmp_vz = (*(dev_txz+tid_txz_0)* *(dev_fdc)
      + *(dev_txz+tid_txz_1)* *(dev_fdc+1)
      + *(dev_txz+tid_txz_2)* *(dev_fdc+2)
      + *(dev_txz+tid_txz_3)* *(dev_fdc+3)) / dx;

      tmp_vz = tmp_vz + (*(dev_tyz+tid_tyz_0)* *(dev_fdc)
      + *(dev_tyz+tid_tyz_1)* *(dev_fdc+1)
      + *(dev_tyz+tid_tyz_2)* *(dev_fdc+2)
      + *(dev_tyz+tid_tyz_3)* *(dev_fdc+3)) / dy;

      tmp_vz = tmp_vz + (*(dev_tzz+tid_tzz_0)* *(dev_fdc)
      + *(dev_tzz+tid_tzz_1)* *(dev_fdc+1)
      + *(dev_tzz+tid_tzz_2)* *(dev_fdc+2)
      + *(dev_tzz+tid_tzz_3)* *(dev_fdc+3)) / dz;

      tmp_rho_vz = 2/(*(dev_rho + tid_rho_vz_0)
      + *(dev_rho + tid_rho_vz_1));

      *(dev_vz+tid_vz) = *(dev_vz+tid_vz) + tmp_rho_vz * tmp_vz;
    }
  }
  //double *fdc/
  //double *intvl: dz, dx, dt
  //double *modelsize : BDnDZ, BDnHX, nDZ, nHX
  el3d_cump(double *vx, double *tmp_vx, int nx_vx, int ny_vx, int nz_vx, int BD_nx_vx, int BD_ny_vx, int BD_vz_vx,
  double *pvxbtxx, double *pvxbtxy, double *pvxbtxz,
  double *vy, double *tmp_vy, int nx_vy, int ny_vy, int nz_vy, int BD_nx_vy, int BD_ny_vy, int BD_vz_vy,
  double *pvybtxy, double *pvybtyy, double *bvybtyz,
  double *vz, double *tmp_vz, int nx_vz, int ny_vz, int nz_vz, int BD_nx_vz, int BD_ny_vz, int BD_vz_vz,
  double *pvzbtxz, double *pvzbtyz, double *bvzbtzz,
  double *txx, double *tmp_txx,
  double *ptxxbvx, double *ptxxbvy, double *ptxxbvz,
  double *tyy, double *tmp_tyy,
  double *ptyybvx, double *ptyybvy, double *ptyybvz,
  double *tzz, double *tmp_tzz,
  double *ptzzbvx, double *ptzzbvy, double *ptzzbvz,
  int nx_tpp, int ny_tpp, int nz_tpp, int BD_nx_tpp, int BD_ny_tpp, int BD_vz_tpp,
  double *txy, double *tmp_txy, int nx_txy, int ny_txy, int nz_txy, int BD_nx_txy, int BD_ny_txy, int BD_nz_txy,
  double *ptxybvx, double *ptxybvy,
  double *tyz, double *tmp_tyz, int nx_tyz, int ny_tyz, int nz_tyz, int BD_nx_tyz, int BD_ny_tyz, int BD_nz_tyz,
  double *ptyzbvy, double *btyzbvz,
  double *txz, double *tmp_txz, int nx_txz, int ny_txz, int nz_txz, int BD_nx_txz, int BD_ny_txz, int BD_nz_txz,
  double *ptxzbvx, double *btxzbvz,
  double *rho, double dt, double dx, double dy, double dz,
  double *bhalf, double *ahalf, double *bfull, double *afull,
  long int *chk_group_full, long int *chk_group_half, long int Num_group, long int Max_group_dim,
  long int *chk_stream_full, long int *chk_stream_half, long int Max_num_stream, long int Max_stream_dim,
  long int *threadim, long int *blockdim)
  {
    //************************************************//
    //**************** GPU setting *******************//
    //************************************************//
    int gn,sn; // gn : group number; sn : stream number
    hipStream_t stream[Num_stream]; // create streams for GPU
    for(sn=0 ; sn < Num_stream ; sn++){
             hipStreamCreate(&stream[sn]); // create concurrent streams
    }
    hipMalloc((void**)&dev_fdc, 4*sizeof(double)); // copy fdc to device
    hipMemcpy(dev_fdc,fdc,4*sizeof(double),hipMemcpyHostToDevice);
    // number of threads and blocks used
    dim3 threads(*threadim,*(threadim+1),*(threadim+2));
    dim3 blocks(*blockdim,*(blockdim+1),*(blockdim+2));
    // pinned host memory for faster transfer between host and device mem
    double *host_vx, *host_vy, *host_vz;
    double *host_txx; *host_tyy, *host_tzz;
    double *host_txy, *host_tyz, *host_txz;
    double *host_rho, *host_lambda, *host_mu;

    hipHostAlloc((void**)&host_vx, (Max_group_dim+3)*(nx_tpp+3)*(nz_tpp+3)*sizeof(double), hipHostMallocDefault);
    hipHostAlloc((void**)&host_vy, (Max_group_dim+3)*(nx_tpp+3)*(nz_tpp+3)*sizeof(double), hipHostMallocDefault);
    hipHostAlloc((void**)&host_vz, (Max_group_dim+3)*(nx_tpp+3)*(nz_tpp+3)*sizeof(double), hipHostMallocDefault);
    hipHostAlloc((void**)&host_txx, Max_group_dim*(nx_tpp+3)*nz_tpp*sizeof(double), hipHostMallocDefault);
    hipHostAlloc((void**)&host_tyy, (Max_group_dim+3)*nx_tpp*nz_tpp*sizeof(double), hipHostMallocDefault);
    hipHostAlloc((void**)&host_tzz, Max_group_dim*nx_tpp*(nz_tpp+3)*sizeof(double), hipHostMallocDefault);
    hipHostAlloc((void**)&host_txy, (Max_group_dim+3)*(nx_tpp+3)*nz_txy*sizeof(double), hipHostMallocDefault);
    hipHostAlloc((void**)&host_tyz, (Max_group_dim+3)*nx_tpp*(nz_tpp+3)*sizeof(double), hipHostMallocDefault);
    hipHostAlloc((void**)&host_txz, Max_group_dim*(nx_tpp+3)*(nz_tpp+3)*sizeof(double), hipHostMallocDefault);
    hipHostAlloc((void**)&host_rho, (Max_group_dim+1)*nx_tpp*nz_tpp*sizeof(double), hipHostMallocDefault);
    hipHostAlloc((void**)&host_lambda, Max_group_dim*nx_tpp*nz_tpp*sizeof(double), hipHostMallocDefault);
    hipHostAlloc((void**)&host_mu, (Max_group_dim+1)*nx_tpp*nz_tpp*sizeof(double), hipHostMallocDefault);

    // device memory for each stream
    double *dev_vx[Max_num_stream], *dev_vy[Max_num_stream], *dev_vz[Max_num_stream];
    double *dev_txx[Max_num_stream], *dev_tyy[Max_num_stream], *dev_tzz[Max_num_stream];
    double *dev_txy[Max_num_stream], *dev_tyz[Max_num_stream], *dev_txz[Max_num_stream];
    double *dev_rho[Max_num_stream], *dev_lambda[Max_num_stream], *dev_mu[Max_num_stream];

    for(i=0 ; i<Max_num_stream ; i++){
      hipMalloc((void**)&dev_vx[i], Max_stream_dim*(nx_tpp+3)*(nz_tpp+3)*sizeof(double));
      hipMalloc((void**)&dev_vy, (Max_stream_dim+3)*(nx_tpp+3)*(nz_tpp+3)*sizeof(double), hipHostMallocDefault);
      hipMalloc((void**)&dev_vz, (Max_stream_dim+3)*(nx_tpp+3)*(nz_tpp+3)*sizeof(double), hipHostMallocDefault);
      hipMalloc((void**)&dev_txx, Max_stream_dim*(nx_tpp+3)*nz_tpp*sizeof(double), hipHostMallocDefault);
      hipMalloc((void**)&dev_tyy, (Max_stream_dim+3)*nx_tpp*nz_tpp*sizeof(double), hipHostMallocDefault);
      hipMalloc((void**)&dev_tzz, Max_stream_dim*nx_tpp*(nz_tpp+3)*sizeof(double), hipHostMallocDefault);
      hipMalloc((void**)&dev_txy, (Max_stream_dim+3)*(nx_tpp+3)*nz_txy*sizeof(double), hipHostMallocDefault);
      hipMalloc((void**)&dev_tyz, (Max_stream_dim+3)*nx_tpp*(nz_tpp+3)*sizeof(double), hipHostMallocDefault);
      hipMalloc((void**)&dev_txz, Max_stream_dim*(nx_tpp+3)*(nz_tpp+3)*sizeof(double), hipHostMallocDefault);
      hipMalloc((void**)&dev_rho, (Max_stream_dim+1)*nx_tpp*nz_tpp*sizeof(double), hipHostMallocDefault);
      hipMalloc((void**)&dev_lambda, Max_stream_dim*nx_tpp*nz_tpp*sizeof(double), hipHostMallocDefault);
      hipMalloc((void**)&dev_mu, (Max_stream_dim+1)*nx_tpp*nz_tpp*sizeof(double), hipHostMallocDefault);
    }

    //*************************************************//
    // PML boundary is calculated using openmp since   //
    // it is less efficient to create different        //
    // scenario for boundary and inner body part.      //
    //*************************************************//

    //*************** CPU part ***************//
    int i,j,k;
    #pragma omp parallel
    {
      //******************** vxbtxx ********************//
      #pragma omp for collapse(3) nowait
      for(i=0; i<BD_nz_vx; i++){
        for(j=1; j<ext; j++){
          for(k=0; k<BD_ny_vx; k++){
            bound_x(*vx, BD_nz_vx, BD_nx_vx, BD_ny_vx, i, j, k,
              *txx, BD_nz_tpp, BD_nx_tpp, j-1,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+(j+1)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dx, *BDwf, *BDcoeff_b, *BDcoeff_a, ext, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=0; i<ext; i++){
        for(j=ext; j<ext+nx_vx; j++){
          for(k=0; k<BD_ny_vx; k++){
            body_x(*vx, BD_nz_vx, BD_nx_vx, BD_ny_vx, i, j, k,
            *txx, BD_nz_tpp, BD_nx_tpp, j-1,
            0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+(j+1)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
            dx, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext+nz_vx; i<BD_nz_vx; i++){
        for(j=ext; j<ext+nx_vx; j++){
          for(k=0; k<BD_ny_vx; k++){
            body_x(*vx, BD_nz_vx, BD_nx_vx, BD_ny_vx, i, j, k,
              *txx, BD_nz_tpp, BD_nx_tpp, j-1,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+(j+1)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dx, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext; i<ext+nz_vx; i++){
        for(j=ext; j<ext+nx_vx; j++){
          for(k=0; k<ext; k++){
            body_x(*vx, BD_nz_vx, BD_nx_vx, BD_ny_vx, i, j, k,
              *txx, BD_nz_tpp, BD_nx_tpp, j-1,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+(j+1)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dx, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext; i<ext+nz_vx; i++){
        for(j=ext; j<ext+nx_vx; j++){
          for(k=ext+ny_vx; k<2*ext+ny_vx; k++){
            body_x(*vx, BD_nz_vx, BD_nx_vx, BD_ny_vx, i, j, k,
              *txx, BD_nz_tpp, BD_nx_tpp, j-1,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+(j+1)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dx, *fdc);
          }
        }
      }

      //******************* vxbtxy ********************//
      #pragma omp for collapse(3) nowait
      for(i=0; i<BD_nz_vx; i++){
        for(j=0; j<BD_nx_vx; j++){
          for(k=2; k<ext; k++){
            bound_y(*vx, BD_nz_vx, BD_nx_vx, BD_ny_vx, i, j, k,
                *txy, BD_nz_txy, BD_nx_txy, k-2,
                0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+(j+1)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
                dy, *BDwf, *BDcoeff_b, *BDcoeff_a, ext, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=0; i<ext; i++){
        for(j=0; j<BD_nx_vx; j++){
          for(k=ext; k<ext+ny_vx; k++){
            body_y(*vx, BD_nz_vx, BD_nx_vx, BD_ny_vx, i, j, k,
              *txy, BD_nz_txy, BD_nx_txy, k-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+(j+1)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dy, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext+nz_vx; i<BD_nz_vx; i++){
        for(j=0; j<BD_nx_vx; j++){
          for(k=ext; k<ext+ny_vx; k++){
            body_y(*vx, BD_nz_vx, BD_nx_vx, BD_ny_vx, i, j, k,
              *txy, BD_nz_txy, BD_nx_txy, k-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+(j+1)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dy, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext; i<ext+nz_vx; i++){
        for(j=0; j<ext; j++){
          for(k=ext; k<ext+ny_vx; k++){
            body_y(*vx, BD_nz_vx, BD_nx_vx, BD_ny_vx, i, j, k,
              *txy, BD_nz_txy, BD_nx_txy, k-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+(j+1)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dy, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext; i<ext+nz_vx; i++){
        for(j=ext+nx_vx; j<BD_nx_vx; j++){
          for(k=ext; k<ext+ny_vx; k++){
            body_y(*vx, BD_nz_vx, BD_nx_vx, BD_ny_vx, i, j, k,
              *txy, BD_nz_txy, BD_nx_txy, k-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+（j+1）* BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dy, *fdc);
          }
        }
      }

      //******************* vxbtxz ********************//
      #pragma omp for collapse(3) nowait
      for(i=2; i<ext; i++){
        for(j=0; j<BD_nx_vx; j++){
          for(k=0; k<BD_ny_vx; k++){
            bound_z(*vx, BD_nz_vx, BD_nx_vx, BD_ny_vx, i, j, k,
              *txz, BD_nz_txz, BD_nx_txz, i-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+(j+1)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dz, *BDwf, *BDcoeff_b, *BDcoeff_a, ext, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext; i<ext+nz_vx; i++){
        for(j=0; j<BD_nx_vx; j++){
          for(k=0; k<ext; k++){
            body_z(*vx, BD_nz_vx, BD_nx_vx, BD_ny_vx, i, j, k,
              *txz, BD_nz_txz, BD_nx_txz, i-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+(1+j)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dz, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext; i<ext+nz_vx; i++){
        for(j=0; j<BD_nx_vx; j++){
          for(k=ext+ny_vx; k<BD_ny_vx; k++){
            body_z(*vx, BD_nz_vx, BD_nx_vx, BD_ny_vx, i, j, k,
              *txz, BD_nz_txz, BD_nx_txz, i-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+(1+j)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dz, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext; i<ext+nz_vx; i++){
        for(j=0; j<ext; j++){
          for(k=ext; k<ext+ny_vx; k++){
            body_y(*vx, BD_nz_vx, BD_nx_vx, BD_ny_vx, i, j, k,
              *txz, BD_nz_txz, BD_nx_txz, j-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+(1+j)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dz, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext; i<ext+nz_vx; i++){
        for(j=ext+nx_vx; j<BD_nx_vx; j++){
          for(k=ext; k<ext+ny_vx; k++){
            body_y(*vx, BD_nz_vx, BD_nx_vx, BD_ny_vx, i, j, k,
            *txz, BD_nz_txz, BD_nx_txz, j-2,
            0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+(1+j)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
            dz, *fdc);
          }
        }
      }

      //******************** vybtxy ********************//
      #pragma omp for collapse(3) nowait
      for(i=0; i<BD_nz_vy; i++){
        for(j=2; j<ext; j++){
          for(k=0; k<BD_ny_vy; k++){
            bound_x(*vy, BD_nz_vy, BD_nx_vy, BD_ny_vy, i, j, k,
              *txy, BD_nz_txy, BD_nx_txy, j-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp)),
              dx, *BDwf, *BDcoeff_b, *BDcoeff_a, ext, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=0; i<ext; i++){
        for(j=ext; j<ext+nx_vy; j++){
          for(k=0; k<BD_ny_vy; k++){
            body_x(*vy, BD_nz_vy, BD_nx_vy, BD_ny_vy, i, j, k,
              *txy, BD_nz_txy, BD_nx_txy, j-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp)),
              dx, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext+nz_vy; i<BD_nz_vy; i++){
        for(j=ext; j<ext+nx_vy; j++){
          for(k=0; k<BD_ny_vy; k++){
            body_x(*vy, BD_nz_vy, BD_nx_vy, BD_ny_vy, i, j, k,
              *txy, BD_nz_txy, BD_nx_txy, j-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+j*BD_nz_tpp+(k+1)BD_nz_tpp*BD_nx_tpp)),
              dx, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext; i<ext+nz_vy; i++){
        for(j=ext; j<ext+nx_vy; j++){
          for(k=0; k<ext; k++){
            body_x(*vy, BD_nz_vy, BD_nx_vy, BD_ny_vy, i, j, k,
              *txy, BD_nz_txy, BD_nx_txy, j-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp)),
              dx, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext; i<ext+nz_vy; i++){
        for(j=ext; j<ext+nx_vy; j++){
          for(k=ext+ny_vy; k<2*ext+ny_vy; k++){
            body_x(*vy, BD_nz_vy, BD_nx_vy, BD_ny_vy, i, j, k,
              *txy, BD_nz_txy, BD_nx_txy, j-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp)),
              dx, *fdc);
          }
        }
      }

      //******************* vybtyy ********************//
      #pragma omp for collapse(3) nowait
      for(i=0; i<BD_nz_vy; i++){
        for(j=0; j<BD_nx_vy; j++){
          for(k=1; k<ext; k++){
            bound_y(*vy, BD_nz_vy, BD_nx_vy, BD_ny_vy, i, j, k,
              *tyy, BD_nz_tpp, BD_nx_tpp, k-1,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp)),
              dy, *BDwf, *BDcoeff_b, *BDcoeff_a, ext, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=0; i<ext; i++){
        for(j=0; j<BD_nx_vy; j++){
          for(k=ext; k<ext+ny_vy; k++){
            body_y(*vy, BD_nz_vy, BD_nx_vy, BD_ny_vy, i, j, k,
              *tyy, BD_nz_tpp, BD_nx_tpp, k-1,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp)),
              dy, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext+nz_vy; i<BD_nz_vy; i++){
        for(j=0; j<BD_nx_vy; j++){
          for(k=ext; k<ext+ny_vy; k++){
            body_y(*vy, BD_nz_vy, BD_nx_vy, BD_ny_vy, i, j, k,
              *tyy, BD_nz_tpp, BD_nx_tpp, k-1,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp)),
              dy, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext; i<ext+nz_vy; i++){
        for(j=0; j<ext; j++){
          for(k=ext; k<ext+ny_vy; k++){
            body_y(*vy, BD_nz_vy, BD_nx_vy, BD_ny_vy, i, j, k,
              *tyy, BD_nz_tpp, BD_nx_tpp, k-1,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp)),
              dy, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext; i<ext+nz_vy; i++){
        for(j=ext+nx_vy; j<BD_nx_vy; j++){
          for(k=ext; k<ext+ny_vy; k++){
            body_y(*vy, BD_nz_vy, BD_nx_vy, BD_ny_vy, i, j, k,
              *tyy, BD_nz_tpp, BD_nx_tpp, k-1,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+j*BD_nz_tpp+（k+1）*BD_nz_tpp*BD_nx_tpp)),
              dy, *fdc);
          }
        }
      }

      //******************* vybtyz ********************//
      #pragma omp for collapse(3) nowait
      for(i=2; i<ext; i++){
        for(j=0; j<BD_nx_vy; j++){
          for(k=0; k<BD_ny_vy; k++){
            bound_z(*vy, BD_nz_vy, BD_nx_vy, BD_ny_vy, i, j, k,
              *tyz, BD_nz_tyz, BD_nx_tyz, i-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp)),
              dz, *BDwf, *BDcoeff_b, *BDcoeff_a, ext, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext; i<ext+nz_vy; i++){
        for(j=0; j<BD_nx_vy; j++){
          for(k=0; k<ext; k++){
            body_z(*vy, BD_nz_vy, BD_nx_vy, BD_ny_vy, i, j, k,
              *tyz, BD_nz_tyz, BD_nx_tyz, i-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp)),
              dz, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext; i<ext+nz_vy; i++){
        for(j=0; j<BD_nx_vy; j++){
          for(k=ext+ny_vy; k<BD_ny_vy; k++){
            body_z(*vy, BD_nz_vy, BD_nx_vy, BD_ny_vy, i, j, k,
              *tyz, BD_nz_tyz, BD_nx_tyz, i-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp)),
              dz, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext; i<ext+nz_vy; i++){
        for(j=0; j<ext; j++){
          for(k=ext; k<ext+ny_vy; k++){
            body_y(*vy, BD_nz_vy, BD_nx_vy, BD_ny_vy, i, j, k,
              *tyz, BD_nz_tyz, BD_nx_tyz, j-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp)),
              dz, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext; i<ext+nz_vy; i++){
        for(j=ext+nx_vy; j<BD_nx_vy; j++){
          for(k=ext; k<ext+ny_vy; k++){
            body_y(*vy, BD_nz_vy, BD_nx_vy, BD_ny_vy, i, j, k,
              *tyz, BD_nz_tyz, BD_nx_tyz, j-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+i+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp)),
              dz, *fdc);
          }
        }
      }

      //******************** vzbtxz ********************//
      #pragma omp for collapse(3) nowait
      for(i=0; i<BD_nz_vz; i++){
        for(j=2; j<ext; j++){
          for(k=0; k<BD_ny_vz; k++){
            bound_x(*vz, BD_nz_vz, BD_nx_vz, BD_ny_vz, i, j, k,
              *txz, BD_nz_txz, BD_nx_txz, j-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+(i+1)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dx, *BDwf, *BDcoeff_b, *BDcoeff_a, ext, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=0; i<ext; i++){
        for(j=ext; j<ext+nx_vz; j++){
          for(k=0; k<BD_ny_vz; k++){
            body_x(*vz, BD_nz_vz, BD_nx_vz, BD_ny_vz, i, j, k,
              *txz, BD_nz_txz, BD_nx_txz, j-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+(i+1)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dx, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext+nz_vz; i<BD_nz_vz; i++){
        for(j=ext; j<ext+nx_vz; j++){
          for(k=0; k<BD_ny_vz; k++){
            body_x(*vz, BD_nz_vz, BD_nx_vz, BD_ny_vz, i, j, k,
              *txz, BD_nz_txz, BD_nx_txz, j-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+(i+1)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dx, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext; i<ext+nz_vz; i++){
        for(j=ext; j<ext+nx_vz; j++){
          for(k=0; k<ext; k++){
            body_x(*vy, BD_nz_vz, BD_nx_vz, BD_ny_vz, i, j, k,
              *txz, BD_nz_txz, BD_nx_txz, j-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+(i+1)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dx, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext; i<ext+nz_vz; i++){
        for(j=ext; j<ext+nx_vz; j++){
          for(k=ext+ny_vz; k<2*ext+ny_vz; k++){
            body_x(*vz, BD_nz_vz, BD_nx_vz, BD_ny_vz, i, j, k,
              *txz, BD_nz_txz, BD_nx_txz, j-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+(i+1)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dx, *fdc);
          }
        }
      }

      //******************* vzbtyz ********************//
      #pragma omp for collapse(3) nowait
      for(i=0; i<BD_nz_vz; i++){
        for(j=0; j<BD_nx_vz; j++){
          for(k=2; k<ext; k++){
            bound_y(*vz, BD_nz_vz, BD_nx_vz, BD_ny_vz, i, j, k,
              *tyz, BD_nz_tyz, BD_nx_tyz, k-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+(i+1)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dy, *BDwf, *BDcoeff_b, *BDcoeff_a, ext, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=0; i<ext; i++){
        for(j=0; j<BD_nx_vz; j++){
          for(k=ext; k<ext+ny_vz; k++){
            body_y(*vz, BD_nz_vz, BD_nx_vz, BD_ny_vz, i, j, k,
              *tyz, BD_nz_tyz, BD_nx_tyz, k-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+(i+1)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dy, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext+nz_vz; i<BD_nz_vz; i++){
        for(j=0; j<BD_nx_vz; j++){
          for(k=ext; k<ext+ny_vz; k++){
            body_y(*vz, BD_nz_vz, BD_nx_vz, BD_ny_vz, i, j, k,
              *tyz, BD_nz_tyz, BD_nx_tyz, k-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+(i+1)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dy, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext; i<ext+nz_vz; i++){
        for(j=0; j<ext; j++){
          for(k=ext; k<ext+ny_vz; k++){
            body_y(*vz, BD_nz_vz, BD_nx_vz, BD_ny_vz, i, j, k,
              *tyz, BD_nz_tyz, BD_nx_tyz, k-2,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+(i+1)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dy, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext; i<ext+nz_vz; i++){
        for(j=ext+nx_vz; j<BD_nx_vz; j++){
          for(k=ext; k<ext+ny_vz; k++){
            body_y(*vz, BD_nz_vz, BD_nx_vz, BD_ny_vz, i, j, k,
              *tyz, BD_nz_tyz, BD_nx_tyz, k-1,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+(i+1)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dy, *fdc);
          }
        }
      }

      //******************* vzbtzz ********************//
      #pragma omp for collapse(3) nowait
      for(i=1; i<ext; i++){
        for(j=0; j<BD_nx_vz; j++){
          for(k=0; k<BD_ny_vz; k++){
            bound_z(*vz, BD_nz_vz, BD_nx_vz, BD_ny_vz, i, j, k,
              *tzz, BD_nz_tpp, BD_nx_tpp, i-1,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+(i+1)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dz, *BDwf, *BDcoeff_b, *BDcoeff_a, ext, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext; i<ext+nz_vz; i++){
        for(j=0; j<BD_nx_vz; j++){
          for(k=0; k<ext; k++){
            body_z(*vz, BD_nz_vz, BD_nx_vz, BD_ny_vz, i, j, k,
              *tzz, BD_nz_tpp, BD_nx_tpp, i-1,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+(i+1)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dz, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext; i<ext+nz_vz; i++){
        for(j=0; j<BD_nx_vz; j++){
          for(k=ext+ny_vz; k<BD_ny_vz; k++){
            body_z(*vz, BD_nz_vz, BD_nx_vz, BD_ny_vz, i, j, k,
              *tzz, BD_nz_tzz, BD_nx_tzz, i-1,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+(i+1)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dz, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext; i<ext+nz_vz; i++){
        for(j=0; j<ext; j++){
          for(k=ext; k<ext+ny_vz; k++){
            body_y(*vz, BD_nz_vz, BD_nx_vz, BD_ny_vz, i, j, k,
              *tzz, BD_nz_tzz, BD_nx_tzz, j-1,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+(i+1)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dz, *fdc);
          }
        }
      }
      #pragma omp for collapse(3) nowait
      for(i=ext; i<ext+nz_vz; i++){
        for(j=ext+nx_vz; j<BD_nx_vz; j++){
          for(k=ext; k<ext+ny_vz; k++){
            body_y(*vz, BD_nz_vz, BD_nx_vz, BD_ny_vz, i, j, k,
              *tzz, BD_nz_tpp, BD_nx_tpp, j-1,
              0.5*(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(rho+(i+1)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
              dz, *fdc);
          }
        }
      }
      ////////////////// GPU part /////////////////////
      // copy memory to pinned memory with a group
      // group number is larger than 1
      int offset;
      int start_point = 0;
      #pragma omp for nowait
      for(gn=0 ; gn<Num_group; gn++){
        start_point = start_point + *(chk_group_full+gn) + ext;
        offset_full = *(chk_group_full+gn+1);
        offset_half = *(chk_group_half+gn+1);
        // vx, vz, txx, tzz, txz
        for(k=0 ; k<offset_full ; k++){
          // copy vx to pinned memory
          for(j=ext;j<ext+nx_vx;j++){
            memcpy(host_vx+k*nz_vx*nx_vx+(j-ext)*nz_vx, vx+(start_point+k)*(BD_nz_vx*BD_nx_vx)+j*BD_nz_vx+ext, nz_vx*sizeof(double));
          }
          // copy vz to pinned memory
          for(j=ext;j<ext+nx_vz;j++){
            memcpy(host_vz+k*nz_vz*nx_vz+(j-ext)*nz_vx, vz+(start_point+k)*(BD_nz_vz*BD_nx_vz)+j*BD_nz_vz+ext, nz_vz*sizeof(double));
          }
          // copy txx to pinned memory
          for(j=ext-1;j<ext+nx_vx+2;j++){
            memcpy(host_txx+k*nz_tpp*(nx_vx+3)+j*nz_tpp, txx+(start_point+k)*(BD_nz_tpp*BD_nx_tpp)+j*BD_nz_tpp+ext, nz_tpp*sizeof(double));
          }
          // copy tzz to pinned memory
          for(j=ext;j<ext+nx_tpp;j++){
            memcpy(host_tzz+k*(nz_vz+3)*nx_tpp+j*(nz_vz+3), tzz+(start_point+k)*(BD_nz_tpp*BD_nx_tpp)+j*BD_nz_tpp+ext-1, (nz_vz+3)*sizeof(double));
          }
          // copy txz to pinned memory
          for(j=ext-2;j<ext+nx_txz+2,j++){
            memcpy(host_txz+k*(nz_txz+4)*(nx_txz+4), txz+(start_point+k)*(BD_nz_tpp*BD_nx_tpp)+j*BD_nz_tpp+ext-2, (nz_txz+4)*sizeof(double))
            }
          }
        // vy, tyy, txy, tyz
        for(k=0 ; k<offset_half ; k++){
          // copy vy to pinned memory
          for(j=ext;j<ext+nx_vy;j++){
            memcpy(host_vy+k*nz_vy*nx_vy, vy+(start_point+k)*(BD_nz_vy*BD_nx_vy)+j*BD_nz_vy+ext, nz_vy*sizeof(double));
          }
        }
        for(k=0 ; k<offset_half+3; k++){
          // copy tyy to pinned memory
          for(j=ext;j<ext+nx_tpp;j++){
            memcpy(host_tyy+k*nz_tpp*nx_tpp, tyy+(start_point+k-1)*(BD_nz_tpp*BD_nx_tpp)+j*BD_nz_tpp+ext, nz_tpp*sizeof(double));
          }
        }
        for(k=0 ; k<offset_full+3; k++){
          //copy txy to pinned memory
          for(j=ext-2;j<ext+nx_txy+2){
            memcpy(host_txy+k*nz_txy*(nx_txy+4)), txy+(start_point+k-2)*(BD_nz_txy*BD_nx_txy)+j*BD_nz_txy+ext, nz_txy*sizeof(double));
          }
          //copy tyz to pinned memory
          for(j=ext;j<ext+nx_tyz){
            memcpy(host_tyz+k*(nz_tyz+4)*nx_tyz, tyz+(start_point+k-2)*(BD_nz_tyz*BD_nx_tyz)+j*BD_nz_tyz+ext-2, (nz_tyz+4)*sizeof(double));
          }
        }
        for(k=0; k<offset_half+1;k++){
          for(j=ext;j<ext+nx_tpp;j++){
            memcpy(host_rho+k*nz_tpp*nx_tpp,rho+(start_point+k)*(BD_nz_tpp*BD_nx_tpp)+j*BD_nz_tpp+ext, nz_tpp*sizeof(double));
          }
        }

        /// copy host memory data to device memory()
        start_point = 0;
        for(sn=0 ; sn<*(chk_stream_full+gn*(Max_num_stream+2)); sn++){
          start_point = start_point + *(chk_stream_full+gn*(Max_num_stream+2)+sn+1);
          offset = *(chk_stream_full+gn*(Max_num_stream+2)+sn+2);
          hipMemcpyAsync(dev_vx[sn], host_vx+start_point*nz_vx*nx_vx, offset*nz_vx*nx_vx*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
          hipMemcpyAsync(dev_vz[sn], host_vz+start_point*nz_vz*nx_vz, offset*nz_vz*nx_vz*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
          hipMemcpyAsync(dev_txx[sn], host_txx+start_point*nz_tpp*(nx_vx+3), offset*nz_vx*(nx_vx+3)*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
          hipMemcpyAsync(dev_tzz[sn], host_tzz+start_point*(nz_vz+3)*nx_tpp, offset*(nz_vz+3)*nx_vz*sizeof(double), hipMemcpyHostToDevice, stream[sn]);)
          hipMemcpyAsync(dev_txz[sn], host_txz+start_point*(nz_vx+3)*(nx_vz+3), offset*(nz_vx+3)*(nx_vz+3)*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
          hipMemcpyAsync(dev_txy[sn], host_txy+start_point*nz_txy*(nx_vy+3), (offset+3)*nz_vx*(nx_vx+3)*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
          hipMemcpyAsync(dev_tyz[sn], host_tyz+start_point*(nz_vy+3)*nx_tyz, (offset+3)*(nz_vz+3)*nx_tyz*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
        }
        for(sn=0 ; sn<*(chk_stream_half+gn*(Max_num_stream+2)); sn++){
          start_point = start_point + *(chk_stream_half+gn*(Max_num_stream+2)+sn+1);
          offset = *(chk_stream_half+gn*(Max_num_stream+2)+sn+2);
          hipMemcpyAsync(dev_vy[sn], host_vy+start_point*nz_vy*nx_vy, offset*nz_vy*nx_vy*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
          hipMemcpyAsync(dev_tyy[sn], host_tyy+start_point*nz_tpp*nx_tpp, (offset+3)*nz_tpp*nx_tpp*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
        }

        for(sn=0 ; sn<*(chk_stream_full+gn*(Max_num_stream+2)); sn++){
          kernel_v<<<blocks,threads,0,stream[sn]>>>(dev_vx[stream_n], dev_rho_vx[stream_n], dev_txx[stream_n], dev_fdc, nvx1, ntpp1, *(sgmt+3), dx);
        }

        for(stream_n=0 ; stream_n<*(sgmt+2); stream_n++){
          hipMemcpyAsync(host_vx+stream_n**(sgmt+3)*nvx1, dev_vx[stream_n], *(sgmt+3)*nvx1*sizeof(double), hipMemcpyDeviceToHost, stream[stream_n]);

        }
        hipMemcpyAsync(host_vx+(*(sgmt+2)-1)**(sgmt+3)*nvx1, dev_vx[*(sgmt+2)-1], *(sgmt+4)*nvx1*sizeof(double), hipMemcpyDeviceToHost, stream[*(sgmt+2)-1]);


        for(stream_n=0 ; stream_n<*(sgmt+2); stream_n++){
                    hipStreamSynchronize(stream[stream_n]);
        }

        for(k=0 ; k<*(sgmt+1) ; k++){
                    memcpy(vx+(group_n**(sgmt+1)+ext+k)*(nvx1+2*ext)+ext, host_vx+k*nvx1, nvx1*sizeof(double));
        }
      }
}
                // end group
               group_n = *sgmt-1;
              for(k=0 ; k<*(sgmt+4) ; k++){
                               memcpy(host_vx+k*nvx1,vx+(group_n**(sgmt+1)+ext+k)*(nvx1+2*ext)+ext,nvx1*sizeof(double));
                               memcpy(host_rho_vx+k*nvx1,rho_vx+(group_n**(sgmt+1)+ext+k)*(nvx1+2*ext)+ext,nvx1*sizeof(double));
                               memcpy(host_txx+k*ntpp1,txx+(group_n**(sgmt+1)+ext-1+k)*(ntpp1+2*ext)+ext,ntpp1*sizeof(double));
             }
            for(k=*(sgmt+4) ; k<*(sgmt+4)+3 ; k++){
                             memcpy(host_txx+k*ntpp1,txx+(group_n**(sgmt+1)+ext-1+k)*(ntpp1+2*ext)+ext,ntpp1*sizeof(double));
           }
           /// copy host memory data to device memory
           for(stream_n=0 ; stream_n<(*(sgmt+5)-1) ; stream_n++){
                           hipMemcpyAsync(dev_vx[stream_n], host_vx+stream_n**(sgmt+6)*nvx1,*(sgmt+6)*nvx1*sizeof(double), hipMemcpyHostToDevice, stream[stream_n]);
                           hipMemcpyAsync(dev_rho_vx[stream_n], host_rho_vx+stream_n**(sgmt+6)*nvx1,*(sgmt+6)*nvx1*sizeof(double), hipMemcpyHostToDevice, stream[stream_n]);
                           hipMemcpyAsync(dev_txx[stream_n], host_txx+stream_n**(sgmt+6)*ntpp1,(*(sgmt+6)+3)*ntpp1*sizeof(double), hipMemcpyHostToDevice, stream[stream_n]);
         }
        hipMemcpyAsync(dev_vx[*(sgmt+5)-1], host_vx+(*(sgmt+5)-1)**(sgmt+6)*nvx1,*(sgmt+7)*nvx1*sizeof(double), hipMemcpyHostToDevice, stream[*(sgmt+5)-1]);
        hipMemcpyAsync(dev_rho_vx[*(sgmt+5)-1], host_rho_vx+(*(sgmt+5)-1)**(sgmt+6)*nvx1,*(sgmt+7)*nvx1*sizeof(double), hipMemcpyHostToDevice, stream[*(sgmt+5)-1]);
        hipMemcpyAsync(dev_txx[*(sgmt+5)-1], host_txx+(*(sgmt+5)-1)**(sgmt+6)*ntpp1,(*(sgmt+7)+3)*ntpp1*sizeof(double), hipMemcpyHostToDevice, stream[*(sgmt+5)-1]);

        for(stream_n=0 ; stream_n<*(sgmt+5) ; stream_n++){
                         kernel_vx<<<blocks,threads,0,stream[stream_n]>>>(dev_vx[stream_n], dev_rho_vx[stream_n], dev_txx[stream_n], dev_fdc, nvx1, ntpp1, *(sgmt+6), dx);
        }
        for(stream_n=0 ; stream_n<(*(sgmt+5)-1) ; stream_n++){
        hipMemcpyAsync(host_vx+stream_n**(sgmt+6)*nvx1,dev_vx[stream_n], *(sgmt+6)*nvx1*sizeof(double), hipMemcpyDeviceToHost, stream[stream_n]);
        // hipMemcpyAsync(host_rho_vx+stream_n**(sgmt+7)*nvx1, dev_rho_vx[stream_n], *(sgmt+7)*nvx1*sizeof(double), hipMemcpyDeviceToHost, stream[stream_n]);
        // hipMemcpyAsync(host_txx+stream_n**(sgmt+7)*ntpp1, dev_txx[stream_n], (*(sgmt+7)+3)*ntpp1*sizeof(double), hipMemcpyDeviceToHost, stream[stream_n]);
                 }
          hipMemcpyAsync(host_vx+(*(sgmt+5)-1)**(sgmt+6)*nvx1, dev_vx[*(sgmt+5)-1], *(sgmt+7)*nvx1*sizeof(double), hipMemcpyDeviceToHost, stream[*(sgmt+5)-1]);
      // hipMemcpyAsync(host_rho_vx+(*(sgmt+6)-1)**(sgmt+3)*nvx1, dev_rho_vx[*(sgmt+6)-1], *(sgmt+8)*nvx1*sizeof(double), hipMemcpyDeviceToHost, stream[*(sgmt+6)-1]);
      // hipMemcpyAsync(host_txx+(*(sgmt+6)-1)**(sgmt+3)*ntpp1, dev_txx[*(sgmt+6)-1], (*(sgmt+8)+3)*ntpp1*sizeof(double), hipMemcpyDeviceToHost, stream[*(sgmt+6)-1]);

      for(stream_n=0 ; stream_n<*(sgmt+5); stream_n++){
        hipStreamSynchronize(stream[stream_n]);
      }

      for(k=0 ; k<*(sgmt+4) ; k++){
        memcpy(vx+(group_n**(sgmt+1)+ext+k)*(nvx1+2*ext)+ext,host_vx+k*nvx1,nvx1*sizeof(double));
      }
    for(i=0 ; i<*(sgmt+2) ; i++){
      hipFree(dev_vx[i]);
      hipFree(dev_rho_vx[i]);
      hipFree(dev_txx[i]);
      hipStreamDestroy(stream[i]);
    }
    hipHostFree(host_vx);
    hipHostFree(host_rho_vx);
    hipHostFree(host_txx);
  }
}
