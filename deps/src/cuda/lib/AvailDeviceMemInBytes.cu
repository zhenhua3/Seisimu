
#include <hip/hip_runtime.h>

unsigned int AvailDeviceMemInBytes()
{
   size_t AvailMemoryInBytes;
   size_t TotalMemoryInBytes;

   hipMemGetInfo(&AvailMemoryInBytes, &TotalMemoryInBytes);

   return AvailMemoryInBytes;
}
