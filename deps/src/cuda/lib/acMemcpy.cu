
#include <hip/hip_runtime.h>
__host__ void acMemcpyHToDforParticleVel(
  double *dev_vx[], double *vx, int BD_nx_vx, int BD_nz_vx,
  double *dev_vy[], double *vy, int BD_nx_vy, int BD_nz_vy,
  double *dev_vz[], double *vz, int BD_nx_vz, int BD_nz_vz,
  double *dev_tpp[], double *tpp, int BD_nx_tpp, int BD_nz_tpp,
  double *dev_rho[], double *rho,
  int TotalStreamNum, int AssignedStreamNum, hipStream_t stream[],
  int *vx_PV_start, int *vy_PV_start, int *vz_PV_start,
  int *tpp_PV_start, int *rho_PV_start,
  int *vx_PV_offset, int *vy_PV_offset, int *vz_PV_offset,
  int *tpp_PV_offset, int *rho_PV_offset)
{
  int nstream;
  hipError_t err;
  for(nstream=0 ; nstream < TotalStreamNum ; nstream++){

    err = hipMemcpyAsync(dev_vx[nstream%AssignedStreamNum],
      vx+*(vx_PV_start+nstream)*BD_nz_vx*BD_nx_vx,
      *(vx_PV_offset+nstream)*BD_nz_vx*BD_nx_vx*sizeof(double),
      hipMemcpyHostToDevice, stream[nstream%AssignedStreamNum]);
      if(err!= hipSuccess){printf("1 %i %s\n",nstream,hipGetErrorString(err));}

    err = hipMemcpyAsync(dev_vy[nstream%AssignedStreamNum],
      vy+*(vy_PV_start+nstream)*BD_nz_vy*BD_nx_vy,
      *(vy_PV_offset+nstream)*BD_nz_vy*BD_nx_vy*sizeof(double),
      hipMemcpyHostToDevice, stream[nstream%AssignedStreamNum]);
      if(err!= hipSuccess){printf("2 %i %s\n",nstream,hipGetErrorString(err));}

    err = hipMemcpyAsync(dev_vz[nstream%AssignedStreamNum],
      vz+*(vz_PV_start+nstream)*BD_nz_vz*BD_nx_vz,
      *(vz_PV_offset+nstream)*BD_nz_vz*BD_nx_vz*sizeof(double),
      hipMemcpyHostToDevice, stream[nstream%AssignedStreamNum]);
      if(err!= hipSuccess){printf("3 %i %s\n",nstream,hipGetErrorString(err));}

    err = hipMemcpyAsync(dev_rho[nstream%AssignedStreamNum],
      rho+*(rho_PV_start+nstream)*BD_nz_tpp*BD_nx_tpp,
      *(rho_PV_offset+nstream)*BD_nz_tpp*BD_nx_tpp*sizeof(double),
      hipMemcpyHostToDevice, stream[nstream%AssignedStreamNum]);
      if(err!= hipSuccess){printf("4 %i %s\n",nstream,hipGetErrorString(err));}

    err = hipMemcpyAsync(dev_tpp[nstream%AssignedStreamNum],
      tpp+*(tpp_PV_start+nstream)*BD_nz_tpp*BD_nx_tpp,
      *(tpp_PV_offset+nstream)*BD_nz_tpp*BD_nx_tpp*sizeof(double),
      hipMemcpyHostToDevice, stream[nstream%AssignedStreamNum]);
      if(err!= hipSuccess){printf("5 %i %s\n",nstream,hipGetErrorString(err));}


  }
}

__host__ void acMemcpyDToHforParticleVel(
  double *dev_vx[], double *vx, int BD_nx_vx, int BD_nz_vx,
  double *dev_vy[], double *vy, int BD_nx_vy, int BD_nz_vy,
  double *dev_vz[], double *vz, int BD_nx_vz, int BD_nz_vz,
  int TotalStreamNum, int AssignedStreamNum, hipStream_t stream[],
  int *vx_PV_start, int *vy_PV_start, int *vz_PV_start,
  int *vx_PV_offset, int *vy_PV_offset, int *vz_PV_offset)
{
  int nstream;
  hipError_t err;
  for(nstream=0 ; nstream < TotalStreamNum ; nstream++){

    err = hipMemcpyAsync(vx+*(vx_PV_start+nstream)*BD_nz_vx*BD_nx_vx,
      dev_vx[nstream%AssignedStreamNum],
      *(vx_PV_offset+nstream)*BD_nz_vx*BD_nx_vx*sizeof(double),
      hipMemcpyDeviceToHost, stream[nstream%AssignedStreamNum]);
      if(err!= hipSuccess){printf("6 %i %s\n",nstream,hipGetErrorString(err));}

    err = hipMemcpyAsync(vy+*(vy_PV_start+nstream)*BD_nz_vy*BD_nx_vy,
      dev_vy[nstream%AssignedStreamNum],
      *(vy_PV_offset+nstream)*BD_nz_vy*BD_nx_vy*sizeof(double),
      hipMemcpyDeviceToHost, stream[nstream%AssignedStreamNum]);
      if(err!= hipSuccess){printf("7 %i %s\n",nstream,hipGetErrorString(err));}


    err = hipMemcpyAsync(vz+*(vz_PV_start+nstream)*BD_nz_vz*BD_nx_vz,
      dev_vz[nstream%AssignedStreamNum],
      *(vz_PV_offset+nstream)*BD_nz_vz*BD_nx_vz*sizeof(double),
      hipMemcpyDeviceToHost, stream[nstream%AssignedStreamNum]);
      if(err!= hipSuccess){printf("8 %i %s\n",nstream,hipGetErrorString(err));}

  }
}


__host__ void acMemcpyHToDforStress(
  double *dev_vx[], double *vx, int BD_nx_vx, int BD_nz_vx,
  double *dev_vy[], double *vy, int BD_nx_vy, int BD_nz_vy,
  double *dev_vz[], double *vz, int BD_nx_vz, int BD_nz_vz,
  double *dev_tpp[], double *tpp, int BD_nx_tpp, int BD_nz_tpp,
  double *dev_lambda[], double *lambda,
  int TotalStreamNum, int AssignedStreamNum, hipStream_t stream[],
  int *vx_SS_start, int *vy_SS_start, int *vz_SS_start,
  int *tpp_SS_start, int *lambda_SS_start,
  int *vx_SS_offset, int *vy_SS_offset, int *vz_SS_offset,
  int *tpp_SS_offset, int *lambda_SS_offset)
  {
    int nstream;
    hipError_t err;
    for(nstream=0 ; nstream < TotalStreamNum ; nstream++){

    err = hipMemcpyAsync(dev_vx[nstream%AssignedStreamNum],
        vx+*(vx_SS_start+nstream)*BD_nz_vx*BD_nx_vx,
        *(vx_SS_offset+nstream)*BD_nz_vx*BD_nx_vx*sizeof(double),
        hipMemcpyHostToDevice, stream[nstream%AssignedStreamNum]);
        if(err!= hipSuccess){printf("9 %s\n",hipGetErrorString(err));}

    err = hipMemcpyAsync(dev_vy[nstream%AssignedStreamNum],
        vy+*(vy_SS_start+nstream)*BD_nz_vy*BD_nx_vy,
        *(vy_SS_offset+nstream)*BD_nz_vy*BD_nx_vy*sizeof(double),
        hipMemcpyHostToDevice, stream[nstream%AssignedStreamNum]);
        if(err!= hipSuccess){printf("10 %s\n",hipGetErrorString(err));}

    err = hipMemcpyAsync(dev_vz[nstream%AssignedStreamNum],
        vz+*(vz_SS_start+nstream)*BD_nz_vz*BD_nx_vz,
        *(vz_SS_offset+nstream)*BD_nz_vz*BD_nx_vz*sizeof(double),
        hipMemcpyHostToDevice, stream[nstream%AssignedStreamNum]);
        if(err!= hipSuccess){printf("11 %s\n",hipGetErrorString(err));}

    err = hipMemcpyAsync(dev_tpp[nstream%AssignedStreamNum],
        tpp+*(tpp_SS_start+nstream)*BD_nz_tpp*BD_nx_tpp,
        *(tpp_SS_offset+nstream)*BD_nz_tpp*BD_nx_tpp*sizeof(double),
        hipMemcpyHostToDevice, stream[nstream%AssignedStreamNum]);
        if(err!= hipSuccess){printf("12 %s\n",hipGetErrorString(err));}

    err = hipMemcpyAsync(dev_lambda[nstream%AssignedStreamNum],
        lambda+*(tpp_SS_start+nstream)*BD_nz_tpp*BD_nx_tpp,
        *(lambda_SS_offset+nstream)*BD_nz_tpp*BD_nx_tpp*sizeof(double),
        hipMemcpyHostToDevice, stream[nstream%AssignedStreamNum]);
        if(err!= hipSuccess){printf("13 %s\n",hipGetErrorString(err));}
      }
    }



__host__ void acMemcpyDToHforStress(
  double *dev_tpp[], double *tpp, int BD_nx_tpp, int BD_nz_tpp,
  int TotalStreamNum, int AssignedStreamNum, hipStream_t stream[],
  int *tpp_SS_start, int *tpp_SS_offset)
  {
    int nstream;
    hipError_t err;
    for(nstream=0 ; nstream < TotalStreamNum ; nstream++){

    err = hipMemcpyAsync(tpp+*(tpp_SS_start+nstream)*BD_nz_tpp*BD_nx_tpp,
        dev_tpp[nstream%AssignedStreamNum],
        *(tpp_SS_offset+nstream)*BD_nz_tpp*BD_nx_tpp*sizeof(double),
        hipMemcpyDeviceToHost, stream[nstream%AssignedStreamNum]);
        if(err!= hipSuccess){printf("14 %s\n",hipGetErrorString(err));}
      }
    }
