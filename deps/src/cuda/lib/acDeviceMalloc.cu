
#include <hip/hip_runtime.h>
__host__ void acDeviceMalloc(double *dev_vx[], int BD_nx_vx, int BD_nz_vx,
  double *dev_vy[], int BD_nx_vy, int BD_nz_vy,
  double *dev_vz[], int BD_nx_vz, int BD_nz_vz,
  double *dev_tpp[], int BD_nx_tpp, int BD_nz_tpp,
  double *dev_rho[], double *dev_lambda[],
  int AssignedStreamNum, int RegStreamDim)
  {
    int nstream, i;
    hipError_t err[6];
    for(nstream=0;nstream<AssignedStreamNum;nstream++){
      err[0] = hipMalloc((void**)&dev_vx[nstream], (RegStreamDim+3)*BD_nx_vx*BD_nz_vx*sizeof(double));
      err[1] = hipMalloc((void**)&dev_vy[nstream], (RegStreamDim+3)*BD_nx_vy*BD_nz_vy*sizeof(double));
      err[2] = hipMalloc((void**)&dev_vz[nstream], (RegStreamDim+3)*BD_nx_vz*BD_nz_vz*sizeof(double));
      err[3] = hipMalloc((void**)&dev_tpp[nstream], (RegStreamDim+3)*BD_nx_tpp*BD_nz_tpp*sizeof(double));
      err[4] = hipMalloc((void**)&dev_lambda[nstream], (RegStreamDim+3)*BD_nx_tpp*BD_nz_tpp*sizeof(double));
      err[5] = hipMalloc((void**)&dev_rho[nstream], (RegStreamDim+3)*BD_nx_tpp*BD_nz_tpp*sizeof(double));
      for(i=0;i<6;i++)
      {
        if(err[i]!= hipSuccess)
        {printf("Device Memory Allocation Error No. %i: %s\n",i, hipGetErrorString(err[i]));}
      }
    }
  }
