__host__ void acKernelExecforParticleVel(
  double *dev_vx[], double *vx, int BD_nx_vx, int BD_nz_vx,
  double *dev_vy[], double *vy, int BD_nx_vy, int BD_nz_vy,
  double *dev_vz[], double *vz, int BD_nx_vz, int BD_nz_vz,
  double *dev_tpp[], double *tpp, int BD_nx_tpp, int BD_nz_tpp,
  double *dev_rho[], double *rho,
  double dx, double dy, double dz, double dt,
  dim3 blocks, dim3 threads,
  int TotalStreamNum, int AssignedStreamNum, hipStream_t stream[],
  int *vx_PV_offset, int *vy_PV_offset, int *vz_PV_offset,
  int *vx_PV_start, int *vy_PV_start, int *vz_PV_start,
  int *tpp_PV_start, int *rho_PV_start)
  {
    int nstream;
    for(nstream=0; nstream < TotalStreamNum; nstream++){
      ackernel_v<<<blocks,threads,0,stream[nstream%AssignedStreamNum]>>>
        (dev_vx[nstream%AssignedStreamNum], BD_nx_vx, BD_nz_vx,
         dev_vy[nstream%AssignedStreamNum], BD_nx_vy, BD_nz_vy,
         dev_vz[nstream%AssignedStreamNum], BD_nx_vz, BD_nz_vz,
         dev_tpp[nstream%AssignedStreamNum], BD_nx_tpp, BD_nz_tpp,
         dev_rho[nstream%AssignedStreamNum], dx, dy, dz, dt,
         *(vx_PV_offset+nstream),*(vy_PV_offset+nstream),*(vz_PV_offset+nstream),
       *(vx_PV_start+nstream),*(vy_PV_start+nstream),*(vz_PV_start+nstream),
     *(tpp_PV_start+nstream),*(rho_PV_start+nstream));
    }
  }


__host__ void acKernelExecforStress(
  double *dev_vx[], double *vx, int BD_nx_vx, int BD_nz_vx,
  double *dev_vy[], double *vy, int BD_nx_vy, int BD_nz_vy,
  double *dev_vz[], double *vz, int BD_nx_vz, int BD_nz_vz,
  double *dev_tpp[], double *tpp, int BD_nx_tpp, int BD_nz_tpp,
  double *dev_lambda[], double *lambda,
  double dx, double dy, double dz, double dt,
  dim3 blocks, dim3 threads,
  int TotalStreamNum, int AssignedStreamNum, hipStream_t stream[],
  int *tpp_SS_offset)
  {
    int nstream;
    for(nstream=0; nstream < TotalStreamNum; nstream++){
      ackernel_tau<<<blocks,threads,0,stream[nstream%AssignedStreamNum]>>>
        (dev_vx[nstream%AssignedStreamNum], BD_nx_vx, BD_nz_vx,
          dev_vy[nstream%AssignedStreamNum], BD_nx_vy, BD_nz_vy,
          dev_vz[nstream%AssignedStreamNum], BD_nx_vz, BD_nz_vz,
          dev_tpp[nstream%AssignedStreamNum], BD_nx_tpp, BD_nz_tpp,
          dev_lambda[nstream%AssignedStreamNum], dx, dy, dz, dt, *(tpp_SS_offset+nstream));
    }
  }
