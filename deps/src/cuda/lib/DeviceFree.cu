
#include <hip/hip_runtime.h>
__host__ void acDeviceFree(
  double *dev_vx[], double *dev_vy[],
  double *dev_vz[], double *dev_tpp[],
  double *dev_lambda[], double *dev_rho[],
  int AssignedStreamNum, hipStream_t stream[])
  {
    int nstream;
    for(nstream=0 ; nstream<AssignedStreamNum ; nstream++)
    {
      hipFree(dev_vx[nstream]);
      hipFree(dev_vy[nstream]);
      hipFree(dev_vz[nstream]);
      hipFree(dev_tpp[nstream]);
      hipFree(dev_rho[nstream]);
      hipFree(dev_lambda[nstream]);
      hipStreamDestroy(stream[nstream]);
    }
  }
