
#include <hip/hip_runtime.h>
//////////////////// elastic 3D kernel //////////////////
  __global__ void kernel_v(double *dev_vx, int BD_nx_vx, int BD_nz_vx,
    double *dev_vy, int BD_nx_vy, int BD_nz_vy,
    double *dev_vz, int BD_nx_vz, int BD_nz_vz,
    double *dev_txx, double *dev_tyy, double *dev_tzz,
    int BD_nx_tpp, int BD_nz_tpp,
    double *dev_txy, int BD_nx_txy, int BD_nz_txy,
    double *dev_tyz, int BD_nx_tyz, int BD_nz_tyz,
    double *dev_txz, int BD_nx_txz, int BD_nz_txz,
    double *dev_rho, double *dev_fdc,
    double dx, double dy, double dz, double dt,
    int chunk_full, int chunk_half)
  {
    int tX = threadIdx.x + blockIdx.x*blockDim.x;
    int tY = threadIdx.y + blockIdx.y*blockDim.y;
    int tZ = threadIdx.z + blockIdx.z*blockDim.z;

    int tid_vx, tid_vy, tid_vz;
    int tid_tpp_0, tid_tpp_1, tid_tpp_2, tid_tpp_3;
    int tid_txy_0, tid_txy_1, tid_txy_2, tid_txy_3;
    int tid_tyz_0, tid_tyz_1, tid_tyz_2, tid_tyz_3;
    int tid_txz_0, tid_txz_1, tid_txz_2, tid_txz_3;
    int tid_rho_0, tid_rho_1;
    double tmp, tmp_rho;
    //***************** vx ********************//
    tid_vx = tZ + tX*BD_nz_vx + tY*BD_nx_vx*BD_nz_vx;
    tid_rho_0 = tZ + (tX+0)*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
    tid_rho_1 = tZ + (tX+1)*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;

    //***************** vx by txx ********************//
    if(tX < BD_nx_vx-1 && tX > 0 && tY < chunk_full && tZ < BD_nz_vx)
    {
      tmp_rho = (0.5*(*(dev_rho + tid_rho_0) + *(dev_rho + tid_rho_1)))/dt;

      tid_tpp_0 = tZ + (tX-1)*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
      tid_tpp_1 = tZ + (tX+0)*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
      tid_tpp_2 = tZ + (tX+1)*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
      tid_tpp_3 = tZ + (tX+2)*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;

      *(dev_vx+tid_vx) = *(dev_vx+tid_vx)
      + ((*(dev_txx+tid_tpp_0)* *(dev_fdc)
      + *(dev_txx+tid_tpp_1)* *(dev_fdc+1)
      + *(dev_txx+tid_tpp_2)* *(dev_fdc+2)
      + *(dev_txx+tid_tpp_3)* *(dev_fdc+3)) / dx)/ tmp_rho;
    }

    //***************** vx by txy ********************//
    if(tX < BD_nx_vx && tX > 0 && tY < chunk_full && tZ < BD_nz_vx)
    {
      tmp_rho = (0.5*(*(dev_rho + tid_rho_0) + *(dev_rho + tid_rho_1)))/dt;

      tid_txy_0 = tZ + tX*BD_nz_txy + (tY+0)*BD_nx_txy*BD_nz_txy;
      tid_txy_1 = tZ + tX*BD_nz_txy + (tY+1)*BD_nx_txy*BD_nz_txy;
      tid_txy_2 = tZ + tX*BD_nz_txy + (tY+2)*BD_nx_txy*BD_nz_txy;
      tid_txy_3 = tZ + tX*BD_nz_txy + (tY+3)*BD_nx_txy*BD_nz_txy;

      *(dev_vx+tid_vx) = *(dev_vx+tid_vx)
      + ((*(dev_txy+tid_txy_0)* *(dev_fdc)
      + *(dev_txy+tid_txy_1)* *(dev_fdc+1)
      + *(dev_txy+tid_txy_2)* *(dev_fdc+2)
      + *(dev_txy+tid_txy_3)* *(dev_fdc+3)) / dy) / tmp_rho;
    }

    // ***************** vx by txz ********************//
    if(tX < BD_nx_vx && tY < chunk_full && tZ < BD_nz_vx-2 && tZ > 1)
    {
      tmp_rho = (0.5*(*(dev_rho + tid_rho_0) + *(dev_rho + tid_rho_1)))/dt;

      tid_txz_0 = (tZ-2) + tX*BD_nz_txz + tY*BD_nx_txz*BD_nz_txz;
      tid_txz_1 = (tZ-1) + tX*BD_nz_txz + tY*BD_nx_txz*BD_nz_txz;
      tid_txz_2 = (tZ+0) + tX*BD_nz_txz + tY*BD_nx_txz*BD_nz_txz;
      tid_txz_3 = (tZ+1) + tX*BD_nz_txz + tY*BD_nx_txz*BD_nz_txz;

      *(dev_vx+tid_vx) = *(dev_vx+tid_vx)
      + ((*(dev_txz+tid_txz_0)* *(dev_fdc)
      + *(dev_txz+tid_txz_1)* *(dev_fdc+1)
      + *(dev_txz+tid_txz_2)* *(dev_fdc+2)
      + *(dev_txz+tid_txz_3)* *(dev_fdc+3)) / dz) / tmp_rho;
    }

    //***************** vy ********************//
    tid_vy = tZ + tX*BD_nz_vy + tY*BD_nx_vy*BD_nz_vy;
    tid_rho_0 = tZ + tX*BD_nz_tpp + (tY+0)*BD_nx_tpp*BD_nz_tpp;
    tid_rho_1 = tZ + tX*BD_nz_tpp + (tY+1)*BD_nx_tpp*BD_nz_tpp;

    //***************** vy by txy********************//
    if(tX < BD_nx_vy-2 && tX > 1 && tY < chunk_half && tZ < BD_nz_vy)
    {
      tmp_rho = (0.5*(*(dev_rho + tid_rho_0) + *(dev_rho + tid_rho_1)))/dt;

      tid_txy_0 = tZ + (tX-2)*BD_nz_txy + (tY+1)*BD_nx_txy*BD_nz_txy;
      tid_txy_1 = tZ + (tX-1)*BD_nz_txy + (tY+1)*BD_nx_txy*BD_nz_txy;
      tid_txy_2 = tZ + (tX+0)*BD_nz_txy + (tY+1)*BD_nx_txy*BD_nz_txy;
      tid_txy_3 = tZ + (tX+1)*BD_nz_txy + (tY+1)*BD_nx_txy*BD_nz_txy;

      *(dev_vy+tid_vy) = *(dev_vy+tid_vy)
      + ((*(dev_txy+tid_txy_0)* *(dev_fdc)
      + *(dev_txy+tid_txy_1)* *(dev_fdc+1)
      + *(dev_txy+tid_txy_2)* *(dev_fdc+2)
      + *(dev_txy+tid_txy_3)* *(dev_fdc+3)) / dx) / tmp_rho;
    }

    //***************** vy by tyy********************//
    if(tX < BD_nx_vy && tY < chunk_half && tZ < BD_nz_vy)
    {
      tmp_rho = (0.5*(*(dev_rho + tid_rho_0) + *(dev_rho + tid_rho_1)))/dt;

      tid_tpp_0 = tZ + tX*BD_nz_tpp + (tY+0)*BD_nx_tpp*BD_nz_tpp;
      tid_tpp_1 = tZ + tX*BD_nz_tpp + (tY+1)*BD_nx_tpp*BD_nz_tpp;
      tid_tpp_2 = tZ + tX*BD_nz_tpp + (tY+2)*BD_nx_tpp*BD_nz_tpp;
      tid_tpp_3 = tZ + tX*BD_nz_tpp + (tY+3)*BD_nx_tpp*BD_nz_tpp;

      *(dev_vy+tid_vy) = *(dev_vy+tid_vy)
      + ((*(dev_tyy+tid_tpp_0)* *(dev_fdc)
      + *(dev_tyy+tid_tpp_1)* *(dev_fdc+1)
      + *(dev_tyy+tid_tpp_2)* *(dev_fdc+2)
      + *(dev_tyy+tid_tpp_3)* *(dev_fdc+3)) / dy) / tmp_rho;
    }

    //***************** vy by tyz********************//
    if(tX < BD_nx_vy && tY < chunk_half && tZ < BD_nz_vy - 2 && tZ > 1)
    {
      tmp_rho = (0.5*(*(dev_rho + tid_rho_0) + *(dev_rho + tid_rho_1)))/dt;

      tid_tyz_0 = (tZ-2) + tX*BD_nz_tyz + (tY+1)*BD_nx_tyz*BD_nz_tyz;
      tid_tyz_1 = (tZ-1) + tX*BD_nz_tyz + (tY+1)*BD_nx_tyz*BD_nz_tyz;
      tid_tyz_2 = (tZ+0) + tX*BD_nz_tyz + (tY+1)*BD_nx_tyz*BD_nz_tyz;
      tid_tyz_3 = (tZ+1) + tX*BD_nz_tyz + (tY+1)*BD_nx_tyz*BD_nz_tyz;

      tmp = ((*(dev_tyz+tid_tyz_0)* *(dev_fdc)
      + *(dev_tyz+tid_tyz_1)* *(dev_fdc+1)
      + *(dev_tyz+tid_tyz_2)* *(dev_fdc+2)
      + *(dev_tyz+tid_tyz_3)* *(dev_fdc+3)) / dz) / tmp_rho;
      *(dev_vy+tid_vy) = *(dev_vy+tid_vy) + tmp;
    }

    //***************** vz ********************//
    tid_vz = tZ + tX*BD_nz_vz + tY*BD_nx_vz*BD_nz_vz;
    tid_rho_0 = (tZ+0) + tX*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
    tid_rho_1 = (tZ+1) + tX*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;


    //***************** vz by txz ********************//
    if(tX < BD_nx_vz-2 && tX > 1 && tY < chunk_full && tZ < BD_nz_vz)
    {
      tmp_rho = (0.5*(*(dev_rho + tid_rho_0) + *(dev_rho + tid_rho_1)))/dt;

      tid_txz_0 = tZ + (tX-2)*BD_nz_txz + tY*BD_nx_txz*BD_nz_txz;
      tid_txz_1 = tZ + (tX-1)*BD_nz_txz + tY*BD_nx_txz*BD_nz_txz;
      tid_txz_2 = tZ + (tX+0)*BD_nz_txz + tY*BD_nx_txz*BD_nz_txz;
      tid_txz_3 = tZ + (tX+1)*BD_nz_txz + tY*BD_nx_txz*BD_nz_txz;

      tmp = ((*(dev_txz+tid_txz_0)* *(dev_fdc)
      + *(dev_txz+tid_txz_1)* *(dev_fdc+1)
      + *(dev_txz+tid_txz_2)* *(dev_fdc+2)
      + *(dev_txz+tid_txz_3)* *(dev_fdc+3)) / dx) / tmp_rho;
      *(dev_vz+tid_vz) = *(dev_vz+tid_vz) + tmp;
    }

    //***************** vz by tyz ********************//
    if(tX < BD_nx_vz && tY < chunk_full && tZ < BD_nz_vz)
    {
      tmp_rho = (0.5*(*(dev_rho + tid_rho_0) + *(dev_rho + tid_rho_1)))/dt;

      tid_tyz_0 = tZ + tX*BD_nz_tyz + (tY+0)*BD_nx_tyz*BD_nz_tyz;
      tid_tyz_1 = tZ + tX*BD_nz_tyz + (tY+1)*BD_nx_tyz*BD_nz_tyz;
      tid_tyz_2 = tZ + tX*BD_nz_tyz + (tY+2)*BD_nx_tyz*BD_nz_tyz;
      tid_tyz_3 = tZ + tX*BD_nz_tyz + (tY+3)*BD_nx_tyz*BD_nz_tyz;

      tmp = ((*(dev_tyz+tid_tyz_0)* *(dev_fdc)
      + *(dev_tyz+tid_tyz_1)* *(dev_fdc+1)
      + *(dev_tyz+tid_tyz_2)* *(dev_fdc+2)
      + *(dev_tyz+tid_tyz_3)* *(dev_fdc+3)) / dy) / tmp_rho;
      *(dev_vz+tid_vz) = *(dev_vz+tid_vz) + tmp;
    }

    //***************** vz by tzz ********************//
    if(tX < BD_nx_vz && tY < chunk_full && tZ < BD_nz_vz-1 && tZ > 0)
    {
      tmp_rho = (0.5*(*(dev_rho + tid_rho_0) + *(dev_rho + tid_rho_1)))/dt;

      tid_tpp_0 = (tZ-1) + tX*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
      tid_tpp_1 = (tZ+0) + tX*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
      tid_tpp_2 = (tZ+1) + tX*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
      tid_tpp_3 = (tZ+2) + tX*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;

      tmp = ((*(dev_tzz+tid_tpp_0)* *(dev_fdc)
      + *(dev_tzz+tid_tpp_1)* *(dev_fdc+1)
      + *(dev_tzz+tid_tpp_2)* *(dev_fdc+2)
      + *(dev_tzz+tid_tpp_3)* *(dev_fdc+3)) / dz) / tmp_rho;
      *(dev_vz+tid_vz) = *(dev_vz+tid_vz) +  tmp;
    }
  }
  __global__ void kernel_tau(double *dev_vx, int BD_nx_vx, int BD_nz_vx,
    double *dev_vy, int BD_nx_vy, int BD_nz_vy,
    double *dev_vz, int BD_nx_vz, int BD_nz_vz,
    double *dev_txx, double *dev_tyy, double *dev_tzz,
    int BD_nx_tpp, int BD_nz_tpp,
    double *dev_txy, int BD_nx_txy, int BD_nz_txy,
    double *dev_tyz, int BD_nx_tyz, int BD_nz_tyz,
    double *dev_txz, int BD_nx_txz, int BD_nz_txz,
    double *dev_lambda, double *dev_mu, double *dev_fdc,
    double dx, double dy, double dz, double dt,
    int chunk_full, int chunk_half)
    {
      int tX = threadIdx.x + blockIdx.x*blockDim.x;
      int tY = threadIdx.y + blockIdx.y*blockDim.y;
      int tZ = threadIdx.z + blockIdx.z*blockDim.z;

      int tid_tpp, tid_txy, tid_txz, tid_tyz, tid_lambda_tpp, tid_mu_tpp;
      int tid_vx_0, tid_vx_1, tid_vx_2, tid_vx_3;
      int tid_vy_0, tid_vy_1, tid_vy_2, tid_vy_3;
      int tid_vz_0, tid_vz_1, tid_vz_2, tid_vz_3;
      int tid_mu_0, tid_mu_1, tid_mu_2, tid_mu_3;
      double tmp_vx, tmp_vy, tmp_vz, tmp_mu;

      //**************** tau_pp ********************//
      tid_tpp = tZ + tX*BD_nz_tpp + tY*BD_nz_tpp*BD_nx_tpp;
      tid_lambda_tpp = tZ + tX*BD_nz_tpp + tY*BD_nz_tpp*BD_nx_tpp;
      tid_mu_tpp = tZ + tX*BD_nz_tpp + (tY+1)*BD_nz_tpp*BD_nx_tpp;
      //**************** tau_pp by vx ********************//
      if(tX > 1 && tX < BD_nx_tpp-2 && tY < chunk_full && tZ < BD_nz_tpp)
      {
        tid_vx_0 = tZ + (tX-2)*BD_nz_vx + (tY+2)*BD_nz_vx*BD_nx_vx;
        tid_vx_1 = tZ + (tX-1)*BD_nz_vx + (tY+2)*BD_nz_vx*BD_nx_vx;
        tid_vx_2 = tZ + (tX+0)*BD_nz_vx + (tY+2)*BD_nz_vx*BD_nx_vx;
        tid_vx_3 = tZ + (tX+1)*BD_nz_vx + (tY+2)*BD_nz_vx*BD_nx_vx;

        tmp_vx = ((*(dev_vx+tid_vx_0)* *(dev_fdc)
        + *(dev_vx+tid_vx_1)* *(dev_fdc+1)
        + *(dev_vx+tid_vx_2)* *(dev_fdc+2)
        + *(dev_vx+tid_vx_3)* *(dev_fdc+3)) / dx)*dt;

        *(dev_txx+tid_tpp) = *(dev_txx+tid_tpp)
        + (*(dev_lambda+tid_lambda_tpp) + 2* *(dev_mu+tid_mu_tpp)) * tmp_vx;

        *(dev_tyy+tid_tpp) = *(dev_tyy+tid_tpp)
        + *(dev_lambda+tid_lambda_tpp) * tmp_vx;

        *(dev_tzz+tid_tpp) = *(dev_tzz+tid_tpp)
        + *(dev_lambda+tid_lambda_tpp) * tmp_vx;
      }
      //**************** tau_pp by vy ********************//
      if(tX < BD_nx_tpp && tY < chunk_full && tZ < BD_nz_tpp)
      {
        tid_vy_0 = tZ + tX*BD_nz_vy + (tY+0)*BD_nz_vy*BD_nx_vy;
        tid_vy_1 = tZ + tX*BD_nz_vy + (tY+1)*BD_nz_vy*BD_nx_vy;
        tid_vy_2 = tZ + tX*BD_nz_vy + (tY+2)*BD_nz_vy*BD_nx_vy;
        tid_vy_3 = tZ + tX*BD_nz_vy + (tY+3)*BD_nz_vy*BD_nx_vy;

        tmp_vy = ((*(dev_vy+tid_vy_0)* *(dev_fdc)
        + *(dev_vy+tid_vy_1)* *(dev_fdc+1)
        + *(dev_vy+tid_vy_2)* *(dev_fdc+2)
        + *(dev_vy+tid_vy_3)* *(dev_fdc+3)) / dy)*dt;

        *(dev_txx+tid_tpp) = *(dev_txx+tid_tpp)
        + *(dev_lambda+tid_lambda_tpp) * tmp_vy;

        *(dev_tyy+tid_tpp) = *(dev_tyy+tid_tpp)
        + (*(dev_lambda+tid_lambda_tpp) + 2* *(dev_mu+tid_mu_tpp)) * tmp_vy;

        *(dev_tzz+tid_tpp) = *(dev_tzz+tid_tpp)
        + *(dev_lambda+tid_lambda_tpp) * tmp_vy;
      }
      //**************** tau_pp by vz ********************//
      if(tX < BD_nx_tpp && tY < chunk_full && tZ > 1 && tZ < BD_nz_tpp-2)
      {
        tid_vz_0 = (tZ-2) + tX*BD_nz_vz + (tY+2)*BD_nz_vz*BD_nx_vz;
        tid_vz_1 = (tZ-1) + tX*BD_nz_vz + (tY+2)*BD_nz_vz*BD_nx_vz;
        tid_vz_2 = (tZ+0) + tX*BD_nz_vz + (tY+2)*BD_nz_vz*BD_nx_vz;
        tid_vz_3 = (tZ+1) + tX*BD_nz_vz + (tY+2)*BD_nz_vz*BD_nx_vz;

        tmp_vz = ((*(dev_vz+tid_vz_0)* *(dev_fdc)
        + *(dev_vz+tid_vz_1)* *(dev_fdc+1)
        + *(dev_vz+tid_vz_2)* *(dev_fdc+2)
        + *(dev_vz+tid_vz_3)* *(dev_fdc+3)) / dz)*dt;

        *(dev_txx+tid_tpp) = *(dev_txx+tid_tpp)
        + *(dev_lambda+tid_lambda_tpp) * tmp_vz;

        *(dev_tyy+tid_tpp) = *(dev_tyy+tid_tpp)
        + *(dev_lambda+tid_lambda_tpp) * tmp_vz;

        *(dev_tzz+tid_tpp) = *(dev_tzz+tid_tpp)
        + (*(dev_lambda+tid_lambda_tpp) + 2* *(dev_mu+tid_mu_tpp)) * tmp_vz;
      }
      //**************** tau_xy ********************//
      tid_txy = tZ + tX*BD_nz_txy + tY*BD_nz_txy*BD_nx_txy;
      tid_mu_0 = tZ + tX*BD_nz_tpp + tY*BD_nz_tpp*BD_nx_tpp;
      tid_mu_1 = tZ + tX*BD_nz_tpp + (tY+1)*BD_nz_tpp*BD_nx_tpp;
      tid_mu_2 = tZ + (tX+1)*BD_nz_tpp + tY*BD_nz_tpp*BD_nx_tpp;
      tid_mu_3 = tZ + (tX+1)*BD_nz_tpp + (tY+1)*BD_nz_tpp*BD_nx_tpp;

      //**************** tau_xy by vy ********************//
      if(tX > 0 && tX < BD_nx_txy-1 && tY < chunk_half && tZ < BD_nz_txy)
      {
        tmp_mu = (*(dev_mu+tid_mu_0) + *(dev_mu+tid_mu_1)
        + *(dev_mu+tid_mu_2) + *(dev_mu+tid_mu_3))/4;

        tid_vy_0 = tZ + (tX-1)*BD_nz_vy + (tY+1)*BD_nz_vy*BD_nx_vy;
        tid_vy_1 = tZ + (tX+0)*BD_nz_vy + (tY+1)*BD_nz_vy*BD_nx_vy;
        tid_vy_2 = tZ + (tX+1)*BD_nz_vy + (tY+1)*BD_nz_vy*BD_nx_vy;
        tid_vy_3 = tZ + (tX+2)*BD_nz_vy + (tY+1)*BD_nz_vy*BD_nx_vy;

        tmp_vy = ((*(dev_vy+tid_vy_0)* *(dev_fdc)
        + *(dev_vy+tid_vy_1)* *(dev_fdc+1)
        + *(dev_vy+tid_vy_2)* *(dev_fdc+2)
        + *(dev_vy+tid_vy_3)* *(dev_fdc+3)) / dx)*dt;
        *(dev_txy+tid_txy) = *(dev_txy+tid_txy) + tmp_vy * tmp_mu;
      }

      //**************** tau_xy by vx ********************//
      if(tX < BD_nx_txy && tY < chunk_full && tZ < BD_nz_txy)
      {
        tmp_mu = (*(dev_mu+tid_mu_0) + *(dev_mu+tid_mu_1)
        + *(dev_mu+tid_mu_2) + *(dev_mu+tid_mu_3))/4;

        tid_vx_0 = tZ + tX*BD_nz_vx + (tY+0)*BD_nz_vx*BD_nx_vx;
        tid_vx_1 = tZ + tX*BD_nz_vx + (tY+1)*BD_nz_vx*BD_nx_vx;
        tid_vx_2 = tZ + tX*BD_nz_vx + (tY+2)*BD_nz_vx*BD_nx_vx;
        tid_vx_3 = tZ + tX*BD_nz_vx + (tY+3)*BD_nz_vx*BD_nx_vx;

        tmp_vx = ((*(dev_vx+tid_vx_0)* *(dev_fdc)
        + *(dev_vx+tid_vx_1)* *(dev_fdc+1)
        + *(dev_vx+tid_vx_2)* *(dev_fdc+2)
        + *(dev_vx+tid_vx_3)* *(dev_fdc+3)) / dy)*dt;

        *(dev_txy+tid_txy) = *(dev_txy+tid_txy) + tmp_mu * tmp_vx;
      }

      //**************** tau_xz ********************//
      tid_txz = tZ + tX*BD_nz_txz + tY*BD_nz_txz*BD_nx_txz;
      tid_mu_0 = tZ + tX*BD_nz_tpp + tY*BD_nz_tpp*BD_nx_tpp;
      tid_mu_1 = (tZ+1) + tX*BD_nz_tpp + tY*BD_nz_tpp*BD_nx_tpp;
      tid_mu_2 = tZ + (tX+1)*BD_nz_tpp + tY*BD_nz_tpp*BD_nx_tpp;
      tid_mu_3 = (tZ+1) + (tX+1)*BD_nz_tpp + tY*BD_nz_tpp*BD_nx_tpp;

      //**************** tau_xz by vx ********************//
      if(tX < BD_nx_txz && tY < chunk_half && tZ < BD_nz_txz-1 && tZ > 0)
      {
        tmp_mu = (*(dev_mu+tid_mu_0) + *(dev_mu+tid_mu_1)
        + *(dev_mu+tid_mu_2) + *(dev_mu+tid_mu_3))/4;

        tid_vx_0 = (tZ-1) + tX*BD_nz_vx + (tY+2)*BD_nz_vx*BD_nx_vx;
        tid_vx_1 = (tZ+0) + tX*BD_nz_vx + (tY+2)*BD_nz_vx*BD_nx_vx;
        tid_vx_2 = (tZ+1) + tX*BD_nz_vx + (tY+2)*BD_nz_vx*BD_nx_vx;
        tid_vx_3 = (tZ+2) + tX*BD_nz_vx + (tY+2)*BD_nz_vx*BD_nx_vx;

        tmp_vx = ((*(dev_vx+tid_vx_0)* *(dev_fdc)
        + *(dev_vx+tid_vx_1)* *(dev_fdc+1)
        + *(dev_vx+tid_vx_2)* *(dev_fdc+2)
        + *(dev_vx+tid_vx_3)* *(dev_fdc+3)) / dz)*dt;

        *(dev_txz+tid_txz) = *(dev_txz+tid_txz) + tmp_mu * tmp_vx;
      }

      //**************** tau_xz by vz ********************//
      if(tX > 0 && tX < BD_nx_txz-1 && tY < chunk_half && tZ < BD_nz_txz)
      {
        tmp_mu = (*(dev_mu+tid_mu_0) + *(dev_mu+tid_mu_1)
        + *(dev_mu+tid_mu_2) + *(dev_mu+tid_mu_3))/4;

        tid_vz_0 = tZ + (tX-1)*BD_nz_vz + (tY+2)*BD_nz_vz*BD_nx_vz;
        tid_vz_1 = tZ + (tX+0)*BD_nz_vz + (tY+2)*BD_nz_vz*BD_nx_vz;
        tid_vz_2 = tZ + (tX+1)*BD_nz_vz + (tY+2)*BD_nz_vz*BD_nx_vz;
        tid_vz_3 = tZ + (tX+2)*BD_nz_vz + (tY+2)*BD_nz_vz*BD_nx_vz;

        tmp_vz = ((*(dev_vz+tid_vz_0)* *(dev_fdc)
        + *(dev_vz+tid_vz_1)* *(dev_fdc+1)
        + *(dev_vz+tid_vz_2)* *(dev_fdc+2)
        + *(dev_vz+tid_vz_3)* *(dev_fdc+3)) / dx)*dt;

        *(dev_txz+tid_txz) = *(dev_txz+tid_txz) + tmp_mu * tmp_vz;
      }

      //**************** tau_yz ********************//
      tid_tyz = tZ + tX*BD_nz_tyz + tY*BD_nz_tyz*BD_nx_tyz;
      tid_mu_0 = tZ + tX*BD_nz_tpp + tY*BD_nz_tpp*BD_nx_tpp;
      tid_mu_1 = (tZ+1) + tX*BD_nz_tpp + tY*BD_nz_tpp*BD_nx_tpp;
      tid_mu_2 = tZ + tX*BD_nz_tpp + (tY+1)*BD_nz_tpp*BD_nx_tpp;
      tid_mu_3 = (tZ+1) + tX*BD_nz_tpp + (tY+1)*BD_nz_tpp*BD_nx_tpp;

      //**************** tau_yz by vy ********************//
      if(tX < BD_nx_tyz && tY < chunk_full && tZ < BD_nz_tyz-1 && tZ > 0)
      {
        tmp_mu = (*(dev_mu+tid_mu_0) + *(dev_mu+tid_mu_1)
        + *(dev_mu+tid_mu_2) + *(dev_mu+tid_mu_3))/4;

        tid_vy_0 = (tZ-1) + tX*BD_nz_vy + (tY+1)*BD_nz_vy*BD_nx_vy;
        tid_vy_1 = (tZ+0) + tX*BD_nz_vy + (tY+1)*BD_nz_vy*BD_nx_vy;
        tid_vy_2 = (tZ+1) + tX*BD_nz_vy + (tY+1)*BD_nz_vy*BD_nx_vy;
        tid_vy_3 = (tZ+2) + tX*BD_nz_vy + (tY+1)*BD_nz_vy*BD_nx_vy;

        tmp_vy = ((*(dev_vy+tid_vy_0)* *(dev_fdc)
        + *(dev_vy+tid_vy_1)* *(dev_fdc+1)
        + *(dev_vy+tid_vy_2)* *(dev_fdc+2)
        + *(dev_vy+tid_vy_3)* *(dev_fdc+3)) / dz)*dt;

        *(dev_tyz+tid_tyz) = *(dev_tyz+tid_tyz) + tmp_mu * tmp_vy;
      }

      //**************** tau_yz by vz ********************//
      if(tX < BD_nx_tyz && tY < chunk_full && tZ < BD_nz_txz)
      {
        tmp_mu = (*(dev_mu+tid_mu_0) + *(dev_mu+tid_mu_1)
        + *(dev_mu+tid_mu_2) + *(dev_mu+tid_mu_3))/4;

        tid_vz_0 = tZ + tX*BD_nz_vz + (tY+0)*BD_nz_vz*BD_nx_vz;
        tid_vz_1 = tZ + tX*BD_nz_vz + (tY+1)*BD_nz_vz*BD_nx_vz;
        tid_vz_2 = tZ + tX*BD_nz_vz + (tY+2)*BD_nz_vz*BD_nx_vz;
        tid_vz_3 = tZ + tX*BD_nz_vz + (tY+3)*BD_nz_vz*BD_nx_vz;

        tmp_vz = ((*(dev_vz+tid_vz_0)* *(dev_fdc)
        + *(dev_vz+tid_vz_1)* *(dev_fdc+1)
        + *(dev_vz+tid_vz_2)* *(dev_fdc+2)
        + *(dev_vz+tid_vz_3)* *(dev_fdc+3)) / dy)*dt;

        *(dev_tyz+tid_tyz) = *(dev_tyz+tid_tyz) + tmp_mu * tmp_vz;
      }
    }
