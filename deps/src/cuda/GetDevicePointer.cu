
#include <hip/hip_runtime.h>
void acGetDevicePointer(
  double *Host_vx, double *Host_vy, double *Host_vz,
  double *Host_tpp, double *Host_rho, double *Host_lambda,
  double *Dev_vx, double *Dev_vy, double *Dev_vz,
  double *Dev_tpp, double *Dev_rho, double *Dev_lambda)
  {
    int err = hipHostGetDevicePointer((void **)&Dev_vx, (void *)Host_vx,0);
    if(err!= hipSuccess){printf("%s\n","error 9");}
    err = hipHostGetDevicePointer((void **)&Dev_vy, (void *)Host_vy,0);
    if(err!= hipSuccess){printf("%s\n","error 10");}
    err = hipHostGetDevicePointer((void **)&Dev_vz, (void *)Host_vz,0);
    if(err!= hipSuccess){printf("%s\n","error 11");}
    err = hipHostGetDevicePointer((void **)&Dev_tpp, (void *)Host_tpp,0);
    if(err!= hipSuccess){printf("%s\n","error 12");}
    err = hipHostGetDevicePointer((void **)&Dev_rho, (void *)Host_rho,0);
    if(err!= hipSuccess){printf("%s\n","error 13");}
    err = hipHostGetDevicePointer((void **)&Dev_lambda, (void *)Host_lambda,0);
    if(err!= hipSuccess){printf("%s\n","error 14");}
  }
