
#include <hip/hip_runtime.h>
__constant__ double dev_fdc[4];

////////////// acoustic 3D kernel //////////////////
__global__ void ackernel_v(double *dev_vx, int BD_nx_vx, int BD_ny_vx, int BD_nz_vx,
  double *dev_vy, int BD_nx_vy, int BD_ny_vy, int BD_nz_vy,
  double *dev_vz, int BD_nx_vz, int BD_ny_vz, int BD_nz_vz,
  double *dev_tpp, int BD_nx_tpp, int BD_ny_tpp, int BD_nz_tpp,
  double *dev_rho, double dx, double dy, double dz, double dt)
{
  int tX = threadIdx.x + blockIdx.x*blockDim.x;
  int tY = threadIdx.y + blockIdx.y*blockDim.y;
  int tZ = threadIdx.z + blockIdx.z*blockDim.z;

  int tid_vx, tid_vy, tid_vz;
  int tid_tpp_0, tid_tpp_1, tid_tpp_2, tid_tpp_3;
  int tid_rho_0, tid_rho_1;
  double tmp_rho;
  //***************** vx ********************//
  tid_vx = tZ + tX*BD_nz_vx + tY*BD_nx_vx*BD_nz_vx;
  tid_rho_0 = tZ + (tX+0)*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
  tid_rho_1 = tZ + (tX+1)*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;

  //***************** vx by tpp ********************//
  if(tX < BD_nx_vx-1 && tX > 0 && tY < BD_ny_vx && tZ < BD_nz_vx)
  {
    tmp_rho = (0.5*(*(dev_rho + tid_rho_0) + *(dev_rho + tid_rho_1)))/dt;

    tid_tpp_0 = tZ + (tX-1)*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
    tid_tpp_1 = tZ + (tX+0)*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
    tid_tpp_2 = tZ + (tX+1)*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
    tid_tpp_3 = tZ + (tX+2)*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
    //
    *(dev_vx+tid_vx) = *(dev_vx+tid_vx)
    + ((*(dev_tpp+tid_tpp_0)* *(dev_fdc)
    + *(dev_tpp+tid_tpp_1)* *(dev_fdc+1)
    + *(dev_tpp+tid_tpp_2)* *(dev_fdc+2)
    + *(dev_tpp+tid_tpp_3)* *(dev_fdc+3)) / dx)/ tmp_rho;
  }

  //***************** vy ********************//
  tid_vy = tZ + tX*BD_nz_vy + tY*BD_nx_vy*BD_nz_vy;
  tid_rho_0 = tZ + tX*BD_nz_tpp + (tY+0)*BD_nx_tpp*BD_nz_tpp;
  tid_rho_1 = tZ + tX*BD_nz_tpp + (tY+1)*BD_nx_tpp*BD_nz_tpp;

  //***************** vy by tpp********************//
  if(tX < BD_nx_vy && tY > 0 && tY < BD_ny_vy-1 && tZ < BD_nz_vy)
  {
    tmp_rho = (0.5*(*(dev_rho + tid_rho_0) + *(dev_rho + tid_rho_1)))/dt;

    tid_tpp_0 = tZ + tX*BD_nz_tpp + (tY-1)*BD_nx_tpp*BD_nz_tpp;
    tid_tpp_1 = tZ + tX*BD_nz_tpp + (tY+0)*BD_nx_tpp*BD_nz_tpp;
    tid_tpp_2 = tZ + tX*BD_nz_tpp + (tY+1)*BD_nx_tpp*BD_nz_tpp;
    tid_tpp_3 = tZ + tX*BD_nz_tpp + (tY+2)*BD_nx_tpp*BD_nz_tpp;

    *(dev_vy+tid_vy) = *(dev_vy+tid_vy)
    + ((*(dev_tpp+tid_tpp_0)* *(dev_fdc)
    + *(dev_tpp+tid_tpp_1)* *(dev_fdc+1)
    + *(dev_tpp+tid_tpp_2)* *(dev_fdc+2)
    + *(dev_tpp+tid_tpp_3)* *(dev_fdc+3)) / dy) / tmp_rho;
  }

  //***************** vz ********************//
  tid_vz = tZ + tX*BD_nz_vz + tY*BD_nx_vz*BD_nz_vz;
  tid_rho_0 = (tZ+0) + tX*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
  tid_rho_1 = (tZ+1) + tX*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;

  //***************** vz by tpp ********************//
  if(tX < BD_nx_vz && tY < BD_ny_vz && tZ < BD_nz_vz-1 && tZ > 0)
  {
    tmp_rho = (0.5*(*(dev_rho + tid_rho_0) + *(dev_rho + tid_rho_1)))/dt;

    tid_tpp_0 = (tZ-1) + tX*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
    tid_tpp_1 = (tZ+0) + tX*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
    tid_tpp_2 = (tZ+1) + tX*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
    tid_tpp_3 = (tZ+2) + tX*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;

    *(dev_vz+tid_vz) = *(dev_vz+tid_vz) + ((*(dev_tpp+tid_tpp_0)* *(dev_fdc)
    + *(dev_tpp+tid_tpp_1)* *(dev_fdc+1)
    + *(dev_tpp+tid_tpp_2)* *(dev_fdc+2)
    + *(dev_tpp+tid_tpp_3)* *(dev_fdc+3)) / dz) / tmp_rho;

  }
}

__global__ void ackernel_tau(double *dev_vx, int BD_nx_vx, int BD_ny_vx, int BD_nz_vx,
  double *dev_vy, int BD_nx_vy, int BD_ny_vy, int BD_nz_vy,
  double *dev_vz, int BD_nx_vz, int BD_ny_vz, int BD_nz_vz,
  double *dev_tpp, int BD_nx_tpp, int BD_ny_tpp, int BD_nz_tpp,
  double *dev_lambda,
  double dx, double dy, double dz, double dt)
  {
    int tX = threadIdx.x + blockIdx.x*blockDim.x;
    int tY = threadIdx.y + blockIdx.y*blockDim.y;
    int tZ = threadIdx.z + blockIdx.z*blockDim.z;

    int tid_tpp, tid_lambda_tpp;
    int tid_vx_0, tid_vx_1, tid_vx_2, tid_vx_3;
    int tid_vy_0, tid_vy_1, tid_vy_2, tid_vy_3;
    int tid_vz_0, tid_vz_1, tid_vz_2, tid_vz_3;
    double tmp_vx, tmp_vy, tmp_vz;

    //**************** tau_pp ********************//
    tid_tpp = tZ + tX*BD_nz_tpp + tY*BD_nz_tpp*BD_nx_tpp;
    tid_lambda_tpp = tZ + tX*BD_nz_tpp + tY*BD_nz_tpp*BD_nx_tpp;

    //**************** tau_pp by vx ********************//
    if(tX > 1 && tX < BD_nx_tpp-2 && tY < BD_ny_tpp && tZ < BD_nz_tpp)
    {
      tid_vx_0 = tZ + (tX-2)*BD_nz_vx + tY*BD_nz_vx*BD_nx_vx;
      tid_vx_1 = tZ + (tX-1)*BD_nz_vx + tY*BD_nz_vx*BD_nx_vx;
      tid_vx_2 = tZ + (tX+0)*BD_nz_vx + tY*BD_nz_vx*BD_nx_vx;
      tid_vx_3 = tZ + (tX+1)*BD_nz_vx + tY*BD_nz_vx*BD_nx_vx;

      tmp_vx = ((*(dev_vx+tid_vx_0)* *(dev_fdc)
      + *(dev_vx+tid_vx_1)* *(dev_fdc+1)
      + *(dev_vx+tid_vx_2)* *(dev_fdc+2)
      + *(dev_vx+tid_vx_3)* *(dev_fdc+3)) / dx)*dt;

      *(dev_tpp+tid_tpp) = *(dev_tpp+tid_tpp)
      + *(dev_lambda+tid_lambda_tpp) * tmp_vx;
    }
    //**************** tau_pp by vy ********************//
    if(tX < BD_nx_tpp && tY > 1 && tY < BD_ny_tpp-2  && tZ < BD_nz_tpp)
    {
      tid_vy_0 = tZ + tX*BD_nz_vy + (tY-2)*BD_nz_vy*BD_nx_vy;
      tid_vy_1 = tZ + tX*BD_nz_vy + (tY-1)*BD_nz_vy*BD_nx_vy;
      tid_vy_2 = tZ + tX*BD_nz_vy + (tY+0)*BD_nz_vy*BD_nx_vy;
      tid_vy_3 = tZ + tX*BD_nz_vy + (tY+1)*BD_nz_vy*BD_nx_vy;

      tmp_vy = ((*(dev_vy+tid_vy_0)* *(dev_fdc)
      + *(dev_vy+tid_vy_1)* *(dev_fdc+1)
      + *(dev_vy+tid_vy_2)* *(dev_fdc+2)
      + *(dev_vy+tid_vy_3)* *(dev_fdc+3)) / dy)*dt;

      *(dev_tpp+tid_tpp) = *(dev_tpp+tid_tpp)
      + *(dev_lambda+tid_lambda_tpp) * tmp_vy;
    }
    //**************** tau_pp by vz ********************//
    if(tX < BD_nx_tpp && tY < BD_ny_tpp && tZ > 1 && tZ < BD_nz_tpp-2)
    {
      tid_vz_0 = (tZ-2) + tX*BD_nz_vz + tY*BD_nz_vz*BD_nx_vz;
      tid_vz_1 = (tZ-1) + tX*BD_nz_vz + tY*BD_nz_vz*BD_nx_vz;
      tid_vz_2 = (tZ+0) + tX*BD_nz_vz + tY*BD_nz_vz*BD_nx_vz;
      tid_vz_3 = (tZ+1) + tX*BD_nz_vz + tY*BD_nz_vz*BD_nx_vz;

      tmp_vz = ((*(dev_vz+tid_vz_0)* *(dev_fdc)
      + *(dev_vz+tid_vz_1)* *(dev_fdc+1)
      + *(dev_vz+tid_vz_2)* *(dev_fdc+2)
      + *(dev_vz+tid_vz_3)* *(dev_fdc+3)) / dz)*dt;

      *(dev_tpp+tid_tpp) = *(dev_tpp+tid_tpp)
      + *(dev_lambda+tid_lambda_tpp) * tmp_vz;
    }
  }
