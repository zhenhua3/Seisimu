#include "hip/hip_runtime.h"

#include<stdio.h>
#include"fd_cuda_lib.c"

extern "C"{
  __global__ void kernel_v(double *dev_vx, int BD_nx_vx, int BD_nz_vx,
    double *dev_vy, int BD_nx_vy, int BD_nz_vy,
    double *dev_vz, int BD_nx_vz, int BD_nz_vz,
    double *dev_tpp, int BD_nx_tpp, int BD_nz_tpp,
    double *dev_rho, double *dev_fdc,
    double dx, double dy, double dz, double dt,
    int chunk_full, int chunk_half)
  {
    int tX = threadIdx.x + blockIdx.x*blockDim.x;
    int tY = threadIdx.y + blockIdx.y*blockDim.y;
    int tZ = threadIdx.z + blockIdx.z*blockDim.z;

    int tid_vx, tid_vy, tid_vz;
    int tid_tpp_0, tid_tpp_1, tid_tpp_2, tid_tpp_3;
    int tid_rho_0, tid_rho_1;
    double tmp, tmp_rho;
    //***************** vx ********************//
    tid_vx = tZ + tX*BD_nz_vx + tY*BD_nx_vx*BD_nz_vx;
    tid_rho_0 = tZ + (tX+0)*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
    tid_rho_1 = tZ + (tX+1)*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;

    //***************** vx by txx ********************//
    if(tX < BD_nx_vx-1 && tX > 0 && tY < chunk_full && tZ < BD_nz_vx)
    {
      tmp_rho = (0.5*(*(dev_rho + tid_rho_0) + *(dev_rho + tid_rho_1)))/dt;

      tid_tpp_0 = tZ + (tX-1)*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
      tid_tpp_1 = tZ + (tX+0)*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
      tid_tpp_2 = tZ + (tX+1)*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
      tid_tpp_3 = tZ + (tX+2)*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;

      *(dev_vx+tid_vx) = *(dev_vx+tid_vx)
      + ((*(dev_txx+tid_tpp_0)* *(dev_fdc)
      + *(dev_txx+tid_tpp_1)* *(dev_fdc+1)
      + *(dev_txx+tid_tpp_2)* *(dev_fdc+2)
      + *(dev_txx+tid_tpp_3)* *(dev_fdc+3)) / dx)/ tmp_rho;
    }

    //***************** vy ********************//
    tid_vy = tZ + tX*BD_nz_vy + tY*BD_nx_vy*BD_nz_vy;
    tid_rho_0 = tZ + tX*BD_nz_tpp + (tY+0)*BD_nx_tpp*BD_nz_tpp;
    tid_rho_1 = tZ + tX*BD_nz_tpp + (tY+1)*BD_nx_tpp*BD_nz_tpp;

    //***************** vy by tyy********************//
    if(tX < BD_nx_vy && tY < chunk_half && tZ < BD_nz_vy)
    {
      tmp_rho = (0.5*(*(dev_rho + tid_rho_0) + *(dev_rho + tid_rho_1)))/dt;

      tid_tpp_0 = tZ + tX*BD_nz_tpp + (tY+0)*BD_nx_tpp*BD_nz_tpp;
      tid_tpp_1 = tZ + tX*BD_nz_tpp + (tY+1)*BD_nx_tpp*BD_nz_tpp;
      tid_tpp_2 = tZ + tX*BD_nz_tpp + (tY+2)*BD_nx_tpp*BD_nz_tpp;
      tid_tpp_3 = tZ + tX*BD_nz_tpp + (tY+3)*BD_nx_tpp*BD_nz_tpp;

      *(dev_vy+tid_vy) = *(dev_vy+tid_vy)
      + ((*(dev_tyy+tid_tpp_0)* *(dev_fdc)
      + *(dev_tyy+tid_tpp_1)* *(dev_fdc+1)
      + *(dev_tyy+tid_tpp_2)* *(dev_fdc+2)
      + *(dev_tyy+tid_tpp_3)* *(dev_fdc+3)) / dy) / tmp_rho;
    }

    //***************** vz ********************//
    tid_vz = tZ + tX*BD_nz_vz + tY*BD_nx_vz*BD_nz_vz;
    tid_rho_0 = (tZ+0) + tX*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
    tid_rho_1 = (tZ+1) + tX*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;

    //***************** vz by tzz ********************//
    if(tX < BD_nx_vz && tY < chunk_full && tZ < BD_nz_vz-1 && tZ > 0)
    {
      tmp_rho = (0.5*(*(dev_rho + tid_rho_0) + *(dev_rho + tid_rho_1)))/dt;

      tid_tpp_0 = (tZ-1) + tX*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
      tid_tpp_1 = (tZ+0) + tX*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
      tid_tpp_2 = (tZ+1) + tX*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;
      tid_tpp_3 = (tZ+2) + tX*BD_nz_tpp + tY*BD_nx_tpp*BD_nz_tpp;

      tmp = ((*(dev_tzz+tid_tpp_0)* *(dev_fdc)
      + *(dev_tzz+tid_tpp_1)* *(dev_fdc+1)
      + *(dev_tzz+tid_tpp_2)* *(dev_fdc+2)
      + *(dev_tzz+tid_tpp_3)* *(dev_fdc+3)) / dz) / tmp_rho;
      *(dev_vz+tid_vz) = *(dev_vz+tid_vz) +  tmp;
    }
  }
  __global__ void kernel_tau(double *dev_vx, int BD_nx_vx, int BD_nz_vx,
    double *dev_vy, int BD_nx_vy, int BD_nz_vy,
    double *dev_vz, int BD_nx_vz, int BD_nz_vz,
    double *dev_tpp, int BD_nx_tpp, int BD_nz_tpp,
    double *dev_lambda, double *dev_fdc,
    double dx, double dy, double dz, double dt,
    int chunk_full, int chunk_half)
    {
      int tX = threadIdx.x + blockIdx.x*blockDim.x;
      int tY = threadIdx.y + blockIdx.y*blockDim.y;
      int tZ = threadIdx.z + blockIdx.z*blockDim.z;

      int tid_tpp, tid_lambda_tpp;
      int tid_vx_0, tid_vx_1, tid_vx_2, tid_vx_3;
      int tid_vy_0, tid_vy_1, tid_vy_2, tid_vy_3;
      int tid_vz_0, tid_vz_1, tid_vz_2, tid_vz_3;
      double tmp_vx, tmp_vy, tmp_vz;

      //**************** tau_pp ********************//
      tid_tpp = tZ + tX*BD_nz_tpp + tY*BD_nz_tpp*BD_nx_tpp;
      tid_lambda_tpp = tZ + tX*BD_nz_tpp + tY*BD_nz_tpp*BD_nx_tpp;

      //**************** tau_pp by vx ********************//
      if(tX > 1 && tX < BD_nx_tpp-2 && tY < chunk_full && tZ < BD_nz_tpp)
      {
        tid_vx_0 = tZ + (tX-2)*BD_nz_vx + (tY+2)*BD_nz_vx*BD_nx_vx;
        tid_vx_1 = tZ + (tX-1)*BD_nz_vx + (tY+2)*BD_nz_vx*BD_nx_vx;
        tid_vx_2 = tZ + (tX+0)*BD_nz_vx + (tY+2)*BD_nz_vx*BD_nx_vx;
        tid_vx_3 = tZ + (tX+1)*BD_nz_vx + (tY+2)*BD_nz_vx*BD_nx_vx;

        tmp_vx = ((*(dev_vx+tid_vx_0)* *(dev_fdc)
        + *(dev_vx+tid_vx_1)* *(dev_fdc+1)
        + *(dev_vx+tid_vx_2)* *(dev_fdc+2)
        + *(dev_vx+tid_vx_3)* *(dev_fdc+3)) / dx)*dt;

        *(dev_tpp+tid_tpp) = *(dev_tpp+tid_tpp)
        + *(dev_lambda+tid_lambda_tpp) * tmp_vx;
      }
      //**************** tau_pp by vy ********************//
      if(tX < BD_nx_tpp && tY < chunk_full && tZ < BD_nz_tpp)
      {
        tid_vy_0 = tZ + tX*BD_nz_vy + (tY+0)*BD_nz_vy*BD_nx_vy;
        tid_vy_1 = tZ + tX*BD_nz_vy + (tY+1)*BD_nz_vy*BD_nx_vy;
        tid_vy_2 = tZ + tX*BD_nz_vy + (tY+2)*BD_nz_vy*BD_nx_vy;
        tid_vy_3 = tZ + tX*BD_nz_vy + (tY+3)*BD_nz_vy*BD_nx_vy;

        tmp_vy = ((*(dev_vy+tid_vy_0)* *(dev_fdc)
        + *(dev_vy+tid_vy_1)* *(dev_fdc+1)
        + *(dev_vy+tid_vy_2)* *(dev_fdc+2)
        + *(dev_vy+tid_vy_3)* *(dev_fdc+3)) / dy)*dt;

        *(dev_tpp+tid_tpp) = *(dev_tpp+tid_tpp)
        + *(dev_lambda+tid_lambda_tpp) * tmp_vy;
      }
      //**************** tau_pp by vz ********************//
      if(tX < BD_nx_tpp && tY < chunk_full && tZ > 1 && tZ < BD_nz_tpp-2)
      {
        tid_vz_0 = (tZ-2) + tX*BD_nz_vz + (tY+2)*BD_nz_vz*BD_nx_vz;
        tid_vz_1 = (tZ-1) + tX*BD_nz_vz + (tY+2)*BD_nz_vz*BD_nx_vz;
        tid_vz_2 = (tZ+0) + tX*BD_nz_vz + (tY+2)*BD_nz_vz*BD_nx_vz;
        tid_vz_3 = (tZ+1) + tX*BD_nz_vz + (tY+2)*BD_nz_vz*BD_nx_vz;

        tmp_vz = ((*(dev_vz+tid_vz_0)* *(dev_fdc)
        + *(dev_vz+tid_vz_1)* *(dev_fdc+1)
        + *(dev_vz+tid_vz_2)* *(dev_fdc+2)
        + *(dev_vz+tid_vz_3)* *(dev_fdc+3)) / dz)*dt;

        *(dev_tpp+tid_tpp) = *(dev_tpp+tid_tpp)
        + *(dev_lambda+tid_lambda_tpp) * tmp_vz;
      }
    }
  //double *fdc/
  //double *intvl: dz, dx, dt
  //double *modelsize : BDnDZ, BDnHX, nDZ, nHX
  void ac3d_cump(double *vx, int BD_nx_vx, int BD_ny_vx, int BD_nz_vx,
  double *pvxbtpp,
  double *vy, int BD_nx_vy, int BD_ny_vy, int BD_nz_vy,
  double *pvybtpp,
  double *vz, int BD_nx_vz, int BD_ny_vz, int BD_nz_vz,
  double *pvzbtpp,
  double *tpp, double *ptppbvx, double *ptppbvy, double *ptppbvz,
  int BD_nx_tpp, int BD_ny_tpp, int BD_nz_tpp,
  double *rho, double *lambda, double *fdc,
  double dt, double dx, double dy, double dz, int ext,
  double *bhalf, double *ahalf, double *bfull, double *afull,
  long int *chk_group_full, long int *chk_group_half, long int Num_group, long int Max_group_dim,
  long int *chk_stream_full, long int *chk_stream_half, long int Max_num_stream, long int Max_stream_dim,
  long int *threadim, long int *blockdim)
  {
    int i,j,k,t;
    //************************************************//
    //**************** GPU setting *******************//
    //************************************************//
    int gn,sn; // gn : group number; sn : stream number
    hipStream_t stream[Max_num_stream]; // create streams for GPU
    double *dev_fdc;
    hipMalloc((void**)&dev_fdc, 4*sizeof(double)); // copy fdc to device
    hipMemcpy(dev_fdc,fdc,4*sizeof(double),hipMemcpyHostToDevice);

    for(sn=0 ; sn < Max_num_stream ; sn++){
      hipStreamCreate(&stream[sn]); // create concurrent streams
    }

    // number of threads and blocks used
    dim3 threads(*threadim,*(threadim+1),*(threadim+2));
    dim3 blocks(*blockdim,*(blockdim+1),*(blockdim+2));
    // pinned host memory for faster transfer between host and device mem
    double *host_vx, *host_vy, *host_vz;
    double *host_tpp,
    double *host_rho, *host_lambda;
    hipHostAlloc((void**)&host_vx, (Max_group_dim+3)*BD_nx_vx*BD_nz_vx*sizeof(double), hipHostMallocDefault);
    hipHostAlloc((void**)&host_vy, (Max_group_dim+3)*BD_nx_vy*BD_nz_vy*sizeof(double), hipHostMallocDefault);
    hipHostAlloc((void**)&host_vz, (Max_group_dim+3)*BD_nx_vz*BD_nz_vz*sizeof(double), hipHostMallocDefault);
    hipHostAlloc((void**)&host_tpp, Max_group_dim*BD_nx_tpp*BD_nz_tpp*sizeof(double), hipHostMallocDefault);
    hipHostAlloc((void**)&host_rho, (Max_group_dim+1)*BD_nx_tpp*BD_nz_tpp*sizeof(double), hipHostMallocDefault);
    hipHostAlloc((void**)&host_lambda, Max_group_dim*BD_nx_tpp*BD_nz_tpp*sizeof(double), hipHostMallocDefault);

    // device memory for each stream
    double *dev_vx[Max_num_stream], *dev_vy[Max_num_stream], *dev_vz[Max_num_stream];
    double *dev_tpp[Max_num_stream];
    double *dev_rho[Max_num_stream], *dev_lambda[Max_num_stream];

    for(i=0 ; i<Max_num_stream ; i++){
      hipMalloc((void**)&dev_vx[i], (Max_stream_dim+3)*BD_nx_vx*BD_nz_vx*sizeof(double));
      hipMalloc((void**)&dev_vy[i], (Max_stream_dim+3)*BD_nx_vy*BD_nz_vy*sizeof(double));
      hipMalloc((void**)&dev_vz[i], (Max_stream_dim+3)*BD_nx_vz*BD_nz_vz*sizeof(double));
      hipMalloc((void**)&dev_txx[i], Max_stream_dim*BD_nx_tpp*BD_nz_tpp*sizeof(double));
      hipMalloc((void**)&dev_rho[i], (Max_stream_dim+1)*BD_nx_tpp*BD_nz_tpp*sizeof(double));
      hipMalloc((void**)&dev_lambda[i], Max_stream_dim*BD_nx_tpp*BD_nz_tpp*sizeof(double));
    }

    int start_group_full = 0;
    int start_group_half = 0;
    int offset_group_full;
    int offset_group_half;
    int start_stream_full[Max_num_stream];
    int start_stream_half[Max_num_stream];
    int offset_stream_full[Max_num_stream];
    int offset_stream_half[Max_num_stream];
    for(gn=0 ; gn<Num_group; gn++)
    {
      start_stream_full[0] = 0;
      start_stream_half[0] = 0;
      for(sn=0;sn<Max_num_stream;sn++)
      {
        offset_stream_full[sn] = *(chk_stream_full+gn*(Max_num_stream+2)+sn+2);
        offset_stream_half[sn] = *(chk_stream_half+gn*(Max_num_stream+2)+sn+2);
      }
      for(sn=1;sn<Max_num_stream;sn++)
      {
        start_stream_full[sn] = start_stream_full[sn-1] + offset_stream_full[sn-1];
        start_stream_half[sn] = start_stream_half[sn-1] + offset_stream_half[sn-1];
      }
    }

    //************ time iteration ************//
    for(t=0;t<1;t++)
    {
      // *****************************************************************//
      // ********************** GPU particle velocities ******************//
      // *****************************************************************//
      start_group_full = 2;
      start_group_half = 1;
    for(gn=0 ; gn<Num_group; gn++)
    {
      start_group_full = start_group_full + *(chk_group_full+gn);
      start_group_half = start_group_half + *(chk_group_half+gn);
      offset_group_full = *(chk_group_full+gn+1);
      offset_group_half = *(chk_group_half+gn+1);
      // vx, vz, txx, tzz, txz
      memcpy(host_vx, vx+start_group_full*BD_nz_vx*BD_nx_vx, offset_group_full*BD_nz_vx*BD_nx_vx*sizeof(double));
      memcpy(host_vy, vy+start_group_half*BD_nz_vy*BD_nx_vy, offset_group_half*BD_nz_vy*BD_nx_vy*sizeof(double));
      memcpy(host_vz, vz+start_group_full*BD_nz_vz*BD_nx_vz, offset_group_full*BD_nz_vz*BD_nx_vz*sizeof(double));
      memcpy(host_tpp, tpp+start_group_full*BD_nz_tpp*BD_nx_tpp, offset_group_full*BD_nz_tpp*BD_nx_tpp*sizeof(double));
      memcpy(host_rho, rho+start_group_half*BD_nz_tpp*BD_nx_tpp, (offset_group_half+1)*BD_nz_tpp*BD_nx_tpp*sizeof(double));

      // copy host memory data to device memory()
      for(sn=0 ; sn<*(chk_stream_full+gn*(Max_num_stream+2)); sn++)
      {
        hipMemcpyAsync(dev_vx[sn], host_vx+start_stream_full[sn]*BD_nz_vx*BD_nx_vx, offset_stream_full[sn]*BD_nz_vx*BD_nx_vx*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
        hipMemcpyAsync(dev_vz[sn], host_vz+start_stream_full[sn]*BD_nz_vz*BD_nx_vz, offset_stream_full[sn]*BD_nz_vz*BD_nx_vz*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
        hipMemcpyAsync(dev_txx[sn], host_txx+start_stream_full[sn]*BD_nz_tpp*BD_nx_tpp, offset_stream_full[sn]*BD_nz_tpp*BD_nx_tpp*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
        hipMemcpyAsync(dev_tzz[sn], host_tzz+start_stream_full[sn]*BD_nz_tpp*BD_nx_tpp, offset_stream_full[sn]*BD_nz_tpp*BD_nx_tpp*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
        hipMemcpyAsync(dev_txz[sn], host_txz+start_stream_full[sn]*BD_nz_txz*BD_nx_txz, offset_stream_full[sn]*BD_nz_txz*BD_nx_txz*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
        hipMemcpyAsync(dev_txy[sn], host_txy+start_stream_full[sn]*BD_nz_txy*BD_nx_txy, (offset_stream_full[sn]+3)*BD_nz_txy*BD_nx_txy*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
        hipMemcpyAsync(dev_tyz[sn], host_tyz+start_stream_full[sn]*BD_nz_tyz*BD_nx_tyz, (offset_stream_full[sn]+3)*BD_nz_tyz*BD_nx_tyz*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
        hipMemcpyAsync(dev_vy[sn], host_vy+start_stream_half[sn]*BD_nz_vy*BD_nx_vy, offset_stream_half[sn]*BD_nz_vy*BD_nx_vy*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
        hipMemcpyAsync(dev_tyy[sn], host_tyy+start_stream_half[sn]*BD_nz_tpp*BD_nx_tpp, (offset_stream_half[sn]+3)*BD_nz_tpp*BD_nx_tpp*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
        hipMemcpyAsync(dev_rho[sn], host_rho+start_stream_half[sn]*BD_nz_tpp*BD_nx_tpp, (offset_stream_half[sn]+1)*BD_nz_tpp*BD_nx_tpp*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
      }
      // if(err!= hipSuccess){printf("%s\n","error");}
      for(sn=0 ; sn<*(chk_stream_full+gn*(Max_num_stream+2)); sn++)
      {
        kernel_v<<<blocks,threads,0,stream[sn]>>>
        (dev_vx[sn], BD_nx_vx, BD_nz_vx,
          dev_vy[sn], BD_nx_vy, BD_nz_vy,
          dev_vz[sn], BD_nx_vz, BD_nz_vz,
          dev_txx[sn], dev_tyy[sn], dev_tzz[sn], BD_nx_tpp, BD_nz_tpp,
          dev_txy[sn], BD_nx_txy, BD_nz_txy,
          dev_tyz[sn], BD_nx_tyz, BD_nz_tyz,
          dev_txz[sn], BD_nx_txz, BD_nz_txz,
          dev_rho[sn], dev_fdc, dx, dy, dz, dt,
          offset_stream_full[sn],
          offset_stream_half[sn]);
      }
      for(sn=0 ; sn<*(chk_stream_full+gn*(Max_num_stream+2)); sn++)
      {
        hipMemcpyAsync(host_vx+start_stream_full[sn]*BD_nz_vx*BD_nx_vx, dev_vx[sn], offset_stream_full[sn]*BD_nz_vx*BD_nx_vx*sizeof(double), hipMemcpyDeviceToHost, stream[sn]);
        hipMemcpyAsync(host_vz+start_stream_full[sn]*BD_nz_vz*BD_nx_vz, dev_vz[sn], offset_stream_full[sn]*BD_nz_vz*BD_nx_vz*sizeof(double), hipMemcpyDeviceToHost, stream[sn]);
        hipMemcpyAsync(host_vy+start_stream_half[sn]*BD_nz_vy*BD_nx_vy, dev_vy[sn], offset_stream_half[sn]*BD_nz_vy*BD_nx_vy*sizeof(double), hipMemcpyDeviceToHost, stream[sn]);
        hipMemcpyAsync(host_txx+start_stream_full[sn]*BD_nz_tpp*BD_nx_tpp, dev_txx[sn], offset_stream_full[sn]*BD_nz_tpp*BD_nx_tpp*sizeof(double), hipMemcpyDeviceToHost, stream[sn]);
      }
      hipDeviceSynchronize();

      memcpy(vx+start_group_full*BD_nz_vx*BD_nx_vx, host_vx, offset_group_full*BD_nz_vx*BD_nx_vx*sizeof(double));
      memcpy(vy+start_group_half*BD_nz_vy*BD_nx_vy, host_vy, offset_group_half*BD_nz_vy*BD_nx_vy*sizeof(double));
      memcpy(vz+start_group_full*BD_nz_vz*BD_nx_vz, host_vz, offset_group_full*BD_nz_vz*BD_nx_vz*sizeof(double));
      memcpy(txx+start_group_full*BD_nz_tpp*BD_nx_tpp, host_txx, offset_group_full*BD_nz_tpp*BD_nx_tpp*sizeof(double));
    }
    // *************************************************//
    // ******* openmp particle velocity boundary *******//
    // *************************************************//
    // vxbtxx
    for(k=0; k<BD_ny_vx; k++)
    {
      for(i=0; i<BD_nz_vx; i++)
      {
        for(j=1; j<ext; j++)
        {
          bound_x(vx, BD_nz_vx, BD_nx_vx, BD_ny_vx, i, j, k,
            txx, BD_nz_tpp, BD_nx_tpp, 1,
          2.0/(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)+*(rho+i+(j+1)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
          2.0/(*(rho+i+(BD_nx_vx-1-j)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)+*(rho+i+(BD_nx_vx-j)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
          dx, dt, pvxbtxx, bhalf, ahalf, ext, fdc);
        }
      }
    }
    // vybtxy
    for(k=0; k<BD_ny_vy; k++)
    {
      for(i=0; i<BD_nz_vy; i++)
      {
        for(j=2; j<ext; j++)
        {
          bound_x(vy, BD_nz_vy, BD_nx_vy, BD_ny_vy, i, j, k,
            txy, BD_nz_txy, BD_nx_txy, 2,
          2.0/(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)+*(rho+i+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp)),
          2.0/(*(rho+i+(BD_nx_vy-1-j)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)+*(rho+i+(BD_nx_vy-1-j)*BD_nz_tpp+(BD_ny_vy-k)*BD_nz_tpp*BD_nx_tpp)),
          dx, dt, pvybtxy, bfull, afull, ext, fdc);
        }
      }
    }
    // vzbtxz
    for(k=0; k<BD_ny_vz; k++)
    {
      for(i=0; i<BD_nz_vz; i++)
      {
        for(j=2; j<ext; j++)
        {
          bound_x(vz, BD_nz_vz, BD_nx_vz, BD_ny_vz, i, j, k,
            txz, BD_nz_txz, BD_nx_txz, 2,
          2.0/(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)+*(rho+(i+1)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
          2.0/(*(rho+i+(BD_nx_vz-1-j)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)+*(rho+(i+1)+(BD_nx_vz-1-j)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
          dx,dt, pvzbtxz, bfull, afull, ext, fdc);
        }
      }
    }
  //********************* V_Y *********************//
    // vxbtxy
    for(j=0; j<BD_nx_vx; j++)
    {
      for(i=0; i<BD_nz_vx; i++)
      {
        for(k=2; k<ext; k++)
        {
          bound_y(vx, BD_nz_vx, BD_nx_vx, BD_ny_vx, i, j, k,
            txy, BD_nz_txy, BD_nx_txy, 2,
            2.0/(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)+*(rho+i+(j+1)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
            2.0/(*(rho+i+j*BD_nz_tpp+(BD_ny_vx-1-k)*BD_nz_tpp*BD_nx_tpp)+*(rho+i+(j+1)*BD_nz_tpp+(BD_ny_vx-1-k)*BD_nz_tpp*BD_nx_tpp)),
            dy,dt, pvxbtxy, bfull, afull, ext, fdc);
        }
      }
    }
    // vybtyy
    for(j=0; j<BD_nx_vy; j++)
    {
      for(i=0; i<BD_nz_vy; i++)
      {
        for(k=1; k<ext; k++)
        {
          bound_y(vy, BD_nz_vy, BD_nx_vy, BD_ny_vy, i, j, k,
            tyy, BD_nz_tpp, BD_nx_tpp, 1,
            2.0/(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)+*(rho+i+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp)),
            2.0/(*(rho+i+j*BD_nz_tpp+(BD_ny_vy-1-k)*BD_nz_tpp*BD_nx_tpp)+*(rho+i+j*BD_nz_tpp+(BD_ny_vy-k)*BD_nz_tpp*BD_nx_tpp)),
            dy,dt, pvybtyy, bhalf, ahalf, ext, fdc);
        }
      }
    }
    // vzbtyz
    for(j=0; j<BD_nx_vz; j++)
    {
      for(i=0; i<BD_nz_vz; i++)
      {
        for(k=2; k<ext; k++)
        {
          bound_y(vz, BD_nz_vz, BD_nx_vz, BD_ny_vz, i, j, k,
            tyz, BD_nz_tyz, BD_nx_tyz, 2,
            2.0/(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)+*(rho+(i+1)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
            2.0/(*(rho+i+j*BD_nz_tpp+(BD_ny_vz-1-k)*BD_nz_tpp*BD_nx_tpp)+*(rho+(i+1)+j*BD_nz_tpp+(BD_ny_vz-1-k)*BD_nz_tpp*BD_nx_tpp)),
            dy,dt, pvzbtyz, bfull, afull, ext, fdc);
        }
      }
    }

  //********************* V_Z *********************//
    // vxbtxz
    for(j=0; j<BD_nx_vx; j++)
    {
      for(k=0; k<BD_ny_vx; k++)
      {
        for(i=2; i<ext; i++)
        {
          unlimited_bound_z(vx, BD_nz_vx, BD_nx_vx, BD_ny_vx, i, j, k,
            txz, BD_nz_txz, BD_nx_txz, 2,
          2.0/(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)+*(rho+i+(j+1)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
          2.0/(*(rho+(BD_nz_vx-1-i)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)+*(rho+(BD_nz_vx-1-i)+(j+1)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
          dz,dt, pvxbtxz, bfull, afull, ext, fdc);
        }
      }
    }
    // vybtyz
    for(j=0; j<BD_nx_vy; j++)
    {
      for(k=0; k<BD_ny_vy; k++)
      {
        for(i=2; i<ext; i++)
        {
          unlimited_bound_z(vy, BD_nz_vy, BD_nx_vy, BD_ny_vy, i, j, k,
            tyz, BD_nz_tyz, BD_nx_tyz, 2,
            2.0/(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)+*(rho+i+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp)),
            2.0/(*(rho+(BD_nz_vy-1-i)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)+*(rho+(BD_nz_vy-1-i)+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp)),
            dz,dt, pvybtyz, bfull, afull, ext, fdc);
        }
      }
    }
    // vzbtzz
    for(j=0; j<BD_nx_vz; j++)
    {
      for(k=0; k<BD_ny_vz; k++)
      {
        for(i=1;i<ext;i++)
        {
          unlimited_bound_z(vz, BD_nz_vz, BD_nx_vz, BD_ny_vz, i, j, k,
            tzz, BD_nz_tpp, BD_nx_tpp, 1,
            2.0/(*(rho+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)+*(rho+(i+1)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
            2.0/(*(rho+(BD_nz_vz-1-i)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)+*(rho+(BD_nz_vz-i)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)),
            dz,dt, pvzbtzz, bhalf, ahalf, ext, fdc);
        }
      }
    }

    //*****************************************************************//
    //**************************** GPU stress *************************//
    //*****************************************************************//
    start_group_full = 2;
    start_group_half = 1;
    for(gn=0 ; gn<Num_group; gn++)
    {
      start_group_full = start_group_full + *(chk_group_full+gn);
      start_group_half = start_group_half + *(chk_group_half+gn);
      offset_group_full = *(chk_group_full+gn+1);
      offset_group_half = *(chk_group_half+gn+1);
      // vx, vz, txx, tzz, txz
        memcpy(host_txx, txx+start_group_full*BD_nz_tpp*BD_nx_tpp, offset_group_full*BD_nz_tpp*BD_nx_tpp*sizeof(double));
        memcpy(host_tzz, tzz+start_group_full*BD_nz_tpp*BD_nx_tpp, offset_group_full*BD_nz_tpp*BD_nx_tpp*sizeof(double));
        memcpy(host_tyy, tyy+start_group_full*BD_nz_tpp*BD_nx_tpp, offset_group_half*BD_nz_tpp*BD_nx_tpp*sizeof(double));
        memcpy(host_txy, txy+start_group_half*BD_nz_txy*BD_nx_txy, offset_group_half*BD_nz_txy*BD_nx_txy*sizeof(double));
        memcpy(host_tyz, tyz+start_group_half*BD_nz_tyz*BD_nx_tyz, offset_group_half*BD_nz_tyz*BD_nx_tyz*sizeof(double));
        memcpy(host_txz, txz+start_group_full*BD_nz_txz*BD_nx_txz, offset_group_full*BD_nz_txz*BD_nx_txz*sizeof(double));
        memcpy(host_vx, vx+(start_group_half-1)*BD_nz_vx*BD_nx_vx, (offset_group_half+3)*BD_nz_vx*BD_nx_vx*sizeof(double));
        memcpy(host_vy, vy+(start_group_full-2)*BD_nz_vy*BD_nx_vy, (offset_group_full+3)*BD_nz_vy*BD_nx_vy*sizeof(double));
        memcpy(host_vz, vz+(start_group_half-1)*BD_nz_vz*BD_nx_vz, (offset_group_half+3)*BD_nz_vz*BD_nx_vz*sizeof(double));
        memcpy(host_lambda, lambda+start_group_full*BD_nz_tpp*BD_nx_tpp, offset_group_full*BD_nz_tpp*BD_nx_tpp*sizeof(double));
        memcpy(host_mu, mu+start_group_half*BD_nz_tpp*BD_nx_tpp, (offset_group_half+1)*BD_nz_tpp*BD_nx_tpp*sizeof(double));

      // copy host memory data to device memory()
      for(sn=0 ; sn<*(chk_stream_full+gn*(Max_num_stream+2)); sn++)
      {
        hipMemcpyAsync(dev_txx[sn], host_txx+start_stream_full[sn]*BD_nz_tpp*BD_nx_tpp, offset_stream_full[sn]*BD_nz_tpp*BD_nx_tpp*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
        hipMemcpyAsync(dev_tzz[sn], host_tzz+start_stream_full[sn]*BD_nz_tpp*BD_nx_tpp, offset_stream_full[sn]*BD_nz_tpp*BD_nx_tpp*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
        hipMemcpyAsync(dev_tyy[sn], host_tyy+start_stream_full[sn]*BD_nz_tpp*BD_nx_tpp, offset_stream_full[sn]*BD_nz_tpp*BD_nx_tpp*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
        hipMemcpyAsync(dev_txz[sn], host_txz+start_stream_full[sn]*BD_nz_txz*BD_nx_txz, offset_stream_full[sn]*BD_nz_txz*BD_nx_txz*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
        hipMemcpyAsync(dev_txy[sn], host_txy+start_stream_half[sn]*BD_nz_txy*BD_nx_txy, offset_stream_half[sn]*BD_nz_txy*BD_nx_txy*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
        hipMemcpyAsync(dev_tyz[sn], host_tyz+start_stream_half[sn]*BD_nz_tyz*BD_nx_tyz, offset_stream_half[sn]*BD_nz_tyz*BD_nx_tyz*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
        hipMemcpyAsync(dev_vx[sn], host_vx+start_stream_half[sn]*BD_nz_vx*BD_nx_vx, (offset_stream_half[sn]+3)*BD_nz_vx*BD_nx_vx*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
        hipMemcpyAsync(dev_vz[sn], host_vz+start_stream_half[sn]*BD_nz_vz*BD_nx_vz, (offset_stream_half[sn]+3)*BD_nz_vz*BD_nx_vz*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
        hipMemcpyAsync(dev_vy[sn], host_vy+start_stream_full[sn]*BD_nz_vy*BD_nx_vy, (offset_stream_full[sn]+3)*BD_nz_vy*BD_nx_vy*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
        hipMemcpyAsync(dev_lambda[sn], host_lambda+start_stream_full[sn]*BD_nz_tpp*BD_nx_tpp, offset_stream_full[sn]*BD_nz_tpp*BD_nx_tpp*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
        hipMemcpyAsync(dev_mu[sn], host_mu+start_stream_half[sn]*BD_nz_tpp*BD_nx_tpp, (offset_stream_half[sn]+1)*BD_nz_tpp*BD_nx_tpp*sizeof(double), hipMemcpyHostToDevice, stream[sn]);
      }
      for(sn=0 ; sn<*(chk_stream_full+gn*(Max_num_stream+2)); sn++)
      {
        kernel_tau<<<blocks,threads,0,stream[sn]>>>
        (dev_vx[sn], BD_nx_vx, BD_nz_vx,
          dev_vy[sn], BD_nx_vy, BD_nz_vy,
          dev_vz[sn], BD_nx_vz, BD_nz_vz,
          dev_txx[sn], dev_tyy[sn], dev_tzz[sn], BD_nx_tpp, BD_nz_tpp,
          dev_txy[sn], BD_nx_txy, BD_nz_txy,
          dev_tyz[sn], BD_nx_tyz, BD_nz_tyz,
          dev_txz[sn], BD_nx_txz, BD_nz_txz,
          dev_lambda[sn], dev_mu[sn], dev_fdc, dx, dy, dz, dt,
          offset_stream_full[sn],
          offset_stream_half[sn]);
      }
      for(sn=0 ; sn<*(chk_stream_full+gn*(Max_num_stream+2)); sn++)
      {
        hipMemcpyAsync(host_txx+start_stream_full[sn]*BD_nz_tpp*BD_nx_tpp, dev_txx[sn], offset_stream_full[sn]*BD_nz_tpp*BD_nx_tpp*sizeof(double), hipMemcpyDeviceToHost, stream[sn]);
        hipMemcpyAsync(host_tyy+start_stream_full[sn]*BD_nz_tpp*BD_nx_tpp, dev_tyy[sn], offset_stream_full[sn]*BD_nz_tpp*BD_nx_tpp*sizeof(double), hipMemcpyDeviceToHost, stream[sn]);
        hipMemcpyAsync(host_tzz+start_stream_full[sn]*BD_nz_tpp*BD_nx_tpp, dev_tzz[sn], offset_stream_full[sn]*BD_nz_tpp*BD_nx_tpp*sizeof(double), hipMemcpyDeviceToHost, stream[sn]);
        hipMemcpyAsync(host_txz+start_stream_full[sn]*BD_nz_txz*BD_nx_txz, dev_txz[sn], offset_stream_full[sn]*BD_nz_txz*BD_nx_txz*sizeof(double), hipMemcpyDeviceToHost, stream[sn]);
        hipMemcpyAsync(host_txy+start_stream_half[sn]*BD_nz_txy*BD_nx_txy, dev_txy[sn], offset_stream_half[sn]*BD_nz_txy*BD_nx_txy*sizeof(double), hipMemcpyDeviceToHost, stream[sn]);
        hipMemcpyAsync(host_tyz+start_stream_half[sn]*BD_nz_tyz*BD_nx_tyz, dev_tyz[sn], offset_stream_half[sn]*BD_nz_tyz*BD_nx_tyz*sizeof(double), hipMemcpyDeviceToHost, stream[sn]);
      }

      hipDeviceSynchronize();

      memcpy(txx+start_group_full*BD_nz_tpp*BD_nx_tpp, host_txx, offset_group_full*BD_nz_tpp*BD_nx_tpp*sizeof(double));
      memcpy(tzz+start_group_full*BD_nz_tpp*BD_nx_tpp, host_tzz, offset_group_full*BD_nz_tpp*BD_nx_tpp*sizeof(double));
      memcpy(tyy+start_group_full*BD_nz_tpp*BD_nx_tpp, host_tyy, offset_group_full*BD_nz_tpp*BD_nx_tpp*sizeof(double));
      memcpy(txy+start_group_half*BD_nz_txy*BD_nx_txy, host_txy, offset_group_half*BD_nz_txy*BD_nx_txy*sizeof(double));
      memcpy(tyz+start_group_half*BD_nz_tyz*BD_nx_tyz, host_tyz, offset_group_half*BD_nz_tyz*BD_nx_tyz*sizeof(double));
      memcpy(txz+start_group_full*BD_nz_txz*BD_nx_txz, host_txz, offset_group_full*BD_nz_txz*BD_nx_txz*sizeof(double));
    }

  //*************************************************//
  //******* openmp stress boundary *******//
  //*************************************************//
    // #pragma omp for collapse(3)
      for(k=0; k<BD_ny_tpp; k++)
      {
        for(i=0; i<BD_nz_tpp; i++)
        {
          for(j=2; j<ext; j++)
          {
            bound_tpp_x(txx, tyy, tzz, BD_nz_tpp, BD_nx_tpp, BD_ny_tpp,
            i, j, k, vx, BD_nz_vx, BD_nx_vx, 2, lambda, mu, dx, dt,
            ptxxbvx, ptyybvx, ptzzbvx, bhalf, ahalf, ext, fdc);
          }
        }
      }
      // #pragma omp for collapse(3)
      for(k=0; k<BD_ny_txy; k++)
      {
        for(i=0; i<BD_nz_txy; i++)
        {
          for(j=1; j<ext; j++)
          {
            bound_x(txy, BD_nz_txy, BD_nx_txy, BD_ny_txy, i, j, k,
            vy, BD_nz_vy, BD_nx_vy, 1,
            (*(mu+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(mu+i+(j+1)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)
            + *(mu+i+(j+1)*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp) + *(mu+i+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp))/4,
            (*(mu+i+(BD_nx_txy-1-j)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(mu+i+(BD_nx_txy-j)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)
            + *(mu+i+(BD_nx_txy-j)*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp) + *(mu+i+(BD_nx_txy-1-j)*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp))/4,
            dx, dt, ptxybvy, bfull, afull, ext, fdc);
          }
        }
      }
  //     // #pragma omp for collapse(3)
      for(k=0; k<BD_ny_txz; k++)
      {
        for(i=0; i<BD_nz_txz; i++)
        {
          for(j=1; j<ext; j++)
          {
            bound_x(txz, BD_nz_txz, BD_nx_txz, BD_ny_txz, i, j, k,
            vz, BD_nz_vz, BD_nx_vz, 1,
            (*(mu+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(mu+i+(j+1)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)
            + *(mu+(i+1)+(j+1)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(mu+(i+1)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp))/4,
            (*(mu+i+(BD_nx_txz-1-j)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(mu+i+(BD_nx_txz-j)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)
            + *(mu+(i+1)+(BD_nx_txz-j)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(mu+(i+1)+(BD_nx_txz-1-j)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp))/4,
            dx, dt, ptxzbvz, bfull, afull, ext, fdc);
          }
        }
      }
  //
    //************ T_Y ************//
      // #pragma omp for collapse(3)
      for(i=0; i<BD_nz_tpp; i++)
      {
        for(j=0; j<BD_nx_tpp; j++)
        {
          for(k=2; k<ext; k++)
          {
            bound_tpp_y(txx, tyy, tzz, BD_nz_tpp, BD_nx_tpp, BD_ny_tpp,
            i, j, k, vy, BD_nz_vy, BD_nx_vy, 2, lambda, mu, dy, dt,
            ptxxbvy, ptyybvy, ptzzbvy, bhalf, ahalf, ext, fdc);
          }
        }
      }
  //     // #pragma omp for collapse(3)
      for(i=0; i<BD_nz_txy; i++)
      {
        for(j=0; j<BD_nx_txy; j++)
        {
          for(k=1; k<ext; k++)
          {
            bound_y(txy, BD_nz_txy, BD_nx_txy, BD_ny_txy, i, j, k,
            vx, BD_nz_vx, BD_nx_vx, 1,
            (*(mu+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(mu+i+(j+1)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)
            + *(mu+i+(j+1)*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp) + *(mu+i+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp))/4,
            (*(mu+i+j*BD_nz_tpp+(BD_ny_txy-k-1)*BD_nz_tpp*BD_nx_tpp) + *(mu+i+(j+1)*BD_nz_tpp+(BD_ny_txy-k-1)*BD_nz_tpp*BD_nx_tpp)
            + *(mu+i+(j+1)*BD_nz_tpp+(BD_ny_txy-k)*BD_nz_tpp*BD_nx_tpp) + *(mu+i+j*BD_nz_tpp+(BD_ny_txy-k)*BD_nz_tpp*BD_nx_tpp))/4,
            dy, dt, ptxybvx, bfull, afull, ext, fdc);
          }
        }
      }
      for(i=0; i<BD_nz_tyz; i++)
      {
        for(j=0; j<BD_nx_tyz; j++)
        {
          for(k=1; k<ext; k++)
          {
            bound_y(tyz, BD_nz_tyz, BD_nx_tyz, BD_ny_tyz, i, j, k,
            vz, BD_nz_vz, BD_nx_vz, 1,
            (*(mu+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(mu+i+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp)
            + *(mu+(i+1)+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp) + *(mu+(i+1)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp))/4,
            (*(mu+i+j*BD_nz_tpp+(BD_ny_tyz-1-k)*BD_nz_tpp*BD_nx_tpp) + *(mu+i+j*BD_nz_tpp+(BD_ny_tyz-k)*BD_nz_tpp*BD_nx_tpp)
            + *(mu+(i+1)+j*BD_nz_tpp+(BD_ny_tyz-k)*BD_nz_tpp*BD_nx_tpp) + *(mu+(i+1)+j*BD_nz_tpp+(BD_ny_tyz-1-k)*BD_nz_tpp*BD_nx_tpp))/4,
            dy, dt, ptyzbvz, bfull, afull, ext, fdc);
          }
        }
      }
    //************ T_Z ************//
      // #pragma omp for collapse(3)
      for(j=0; j<BD_nx_tpp; j++)
      {
        for(k=0; k<BD_ny_tpp; k++)
        {
          for(i=2; i<ext; i++)
          {
            unlimited_bound_tpp_z(txx, tyy, tzz, BD_nz_tpp, BD_nx_tpp, BD_ny_tpp,
            i, j, k, vz, BD_nz_vz, BD_nx_vz, 2, lambda, mu, dz, dt,
            ptxxbvz, ptyybvz, ptzzbvz, bhalf, ahalf, ext, fdc);
          }
        }
      }
  //     // #pragma omp for collapse(3)
      for(j=0; j<BD_nx_txz; j++)
      {
        for(k=0; k<BD_ny_txz; k++)
        {
          for(i=1; i<ext; i++)
          {
            unlimited_bound_z(txz, BD_nz_txz, BD_nx_txz, BD_ny_txz, i, j, k,
            vx, BD_nz_vx, BD_nx_vx, 1,
            (*(mu+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(mu+i+(j+1)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)
            + *(mu+(i+1)+(j+1)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(mu+(i+1)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp))/4,
            (*(mu+(BD_nz_txz-1-i)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(mu+(BD_nz_txz-1-i)+(j+1)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp)
            + *(mu+(BD_nz_txz-i)+(j+1)*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(mu+(BD_nz_txz-i)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp))/4,
            dz, dt, ptxzbvx, bfull, afull, ext, fdc);
          }
        }
      }
  //     // #pragma omp for collapse(3)
      for(j=0; j<BD_nx_tyz; j++)
      {
        for(k=0; k<BD_ny_tyz; k++)
        {
          for(i=0; i<ext; i++)
          {
            unlimited_bound_z(tyz, BD_nz_tyz, BD_nx_tyz, BD_ny_tyz, i, j, k,
            vy, BD_nz_vy, BD_nx_vy, 1,
            (*(mu+i+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(mu+i+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp)
            + *(mu+(i+1)+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp) + *(mu+(i+1)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp))/4,
            (*(mu+(BD_nz_tyz-1-i)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp) + *(mu+(BD_nz_tyz-1-i)+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp)
            + *(mu+(BD_nz_tyz-i)+j*BD_nz_tpp+(k+1)*BD_nz_tpp*BD_nx_tpp) + *(mu+(BD_nz_tyz-i)+j*BD_nz_tpp+k*BD_nz_tpp*BD_nx_tpp))/4,
            dz, dt, ptyzbvy, bfull, afull, ext, fdc);
          }
        }
      }
  }
  for(i=0 ; i<Max_num_stream ; i++)
    {
      hipFree(dev_vx[i]);
      hipFree(dev_vy[i]);
      hipFree(dev_vz[i]);
      hipFree(dev_txx[i]);
      hipFree(dev_tyy[i]);
      hipFree(dev_tzz[i]);
      hipFree(dev_txy[i]);
      hipFree(dev_tyz[i]);
      hipFree(dev_txz[i]);
      hipFree(dev_rho[i]);
      hipFree(dev_lambda[i]);
      hipFree(dev_mu[i]);
      hipStreamDestroy(stream[i]);
    }
    hipHostFree(host_vx);
    hipHostFree(host_vy);
    hipHostFree(host_vz);
    hipHostFree(host_txx);
    hipHostFree(host_tyy);
    hipHostFree(host_tzz);
    hipHostFree(host_txy);
    hipHostFree(host_tyz);
    hipHostFree(host_txz);
    hipHostFree(host_rho);
    hipHostFree(host_lambda);
    hipHostFree(host_mu);
  }
}
