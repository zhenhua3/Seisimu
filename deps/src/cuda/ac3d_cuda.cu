extern "C"{
  #include<stdio.h>
  #include<unistd.h>
  #include<sys/stat.h>
  #include<sys/mman.h>
  #include<fcntl.h>
  #include<string.h>
  #include<stdlib.h>
  #include"lib/lib.h"

  //double *fdc/
  //double *intvl: dz, dx, dt
  //double *modelsize : BDnDZ, BDnHX, nDZ, nHX
  void ac3d_cuda(
    double *vx, int BD_nx_vx, int BD_ny_vx, int BD_nz_vx, double *pvxbtpp,
    double *vy, int BD_nx_vy, int BD_ny_vy, int BD_nz_vy, double *pvybtpp,
    double *vz, int BD_nx_vz, int BD_ny_vz, int BD_nz_vz, double *pvzbtpp,
    double *tpp, double *ptppbvx, double *ptppbvy, double *ptppbvz,
    int BD_nx_tpp, int BD_ny_tpp, int BD_nz_tpp,
    double *rho, double *lambda, double *fdc,
    int nT, double dt, double dx, double dy, double dz, int ext,
    double *bhalf, double *ahalf, double *bfull, double *afull,
    char *snp_path, long long dim, int intvl,
    long long *threadim, long long *blockdim,
    int AssignedStreamNum, int TotalStreamNum, int RegStreamDim,
    int *vx_PV_start, int *vy_PV_start, int *vz_PV_start,
    int *tpp_PV_start, int *rho_PV_start,
    int *vx_PV_offset, int *vy_PV_offset, int *vz_PV_offset,
    int *tpp_PV_offset, int *rho_PV_offset,
    int *vx_SS_start, int *vy_SS_start, int *vz_SS_start,
    int *tpp_SS_start, int *lambda_SS_start,
    int *vx_SS_offset, int *vy_SS_offset, int *vz_SS_offset,
    int *tpp_SS_offset, int *lambda_SS_offset)
  {

    int t,nstream;
    hipStream_t stream[AssignedStreamNum];
    for(nstream=0 ; nstream < AssignedStreamNum ; nstream++){
      hipStreamCreate(&stream[nstream]); // create concurrent streams
    }

    // double *snp_ptr = mmap_snapshot(snp_path, dim);

    // copy finite difference coefficient to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(dev_fdc),fdc,4*sizeof(double));

    // number of threads and blocks used
    dim3 threads(*threadim,*(threadim+1),*(threadim+2));
    dim3 blocks(*blockdim,*(blockdim+1),*(blockdim+2));

    // pinned host memory for faster transfer between host and device mem
    acHostRegister(vx, BD_nx_vx, BD_ny_vx, BD_nz_vx,
    vy, BD_nx_vy, BD_ny_vy, BD_nz_vy,
    vz, BD_nx_vz, BD_ny_vz, BD_nz_vz,
    tpp, BD_nx_tpp, BD_ny_tpp, BD_nz_tpp,
    rho, lambda);


    // Assign device space for each stream
    double *dev_vx[AssignedStreamNum], *dev_vy[AssignedStreamNum], *dev_vz[AssignedStreamNum];
    double *dev_tpp[AssignedStreamNum];
    double *dev_rho[AssignedStreamNum], *dev_lambda[AssignedStreamNum];

    acDeviceMalloc(dev_vx, BD_nx_vx, BD_nz_vx,
      dev_vy, BD_nx_vy, BD_nz_vy,
      dev_vz, BD_nx_vz, BD_nz_vz,
      dev_tpp, BD_nx_tpp, BD_nz_tpp,
      dev_rho, dev_lambda,
      AssignedStreamNum, RegStreamDim);



    //************ time iteration ************//
    for(t=0;t<nT;t++)
    {
      // Particle velocities part
      // Copy data from host memory to device memory
      acMemcpyHToDforParticleVel(
        dev_vx, vx, BD_nx_vx, BD_nz_vx,
        dev_vy, vy, BD_nx_vy, BD_nz_vy,
        dev_vz, vz, BD_nx_vz, BD_nz_vz,
        dev_tpp, tpp, BD_nx_tpp, BD_nz_tpp,
        dev_rho, rho,
        TotalStreamNum, AssignedStreamNum, stream,
        vx_PV_start, vy_PV_start, vz_PV_start,
        tpp_PV_start, rho_PV_start,
        vx_PV_offset, vy_PV_offset, vz_PV_offset,
        tpp_PV_offset, rho_PV_offset);

      // Kernekl Execution for particle velocities
      acKernelExecforParticleVel(
        dev_vx, vx, BD_nx_vx, BD_nz_vx,
        dev_vy, vy, BD_nx_vy, BD_nz_vy,
        dev_vz, vz, BD_nx_vz, BD_nz_vz,
        dev_tpp, tpp, BD_nx_tpp, BD_nz_tpp,
        dev_rho, rho,
        dx, dy, dz, dt,
        blocks, threads,
        TotalStreamNum, AssignedStreamNum, stream,
        vx_PV_offset, vy_PV_offset, vz_PV_offset,
        vx_PV_start, vy_PV_start, vz_PV_start,
        tpp_PV_start, rho_PV_start);

      // Copy data from device meory to host memory
      acMemcpyDToHforParticleVel(
        dev_vx, vx, BD_nx_vx, BD_nz_vx,
        dev_vy, vy, BD_nx_vy, BD_nz_vy,
        dev_vz, vz, BD_nx_vz, BD_nz_vz,
        TotalStreamNum, AssignedStreamNum, stream,
        vx_PV_start, vy_PV_start, vz_PV_start,
        vx_PV_offset, vy_PV_offset, vz_PV_offset);

      // finish computing particle velocities before computing stress
      hipDeviceSynchronize();

      // Stress part
      // Copy data from host memory to device memory
      acMemcpyHToDforStress(
        dev_vx, vx, BD_nx_vx, BD_nz_vx,
        dev_vy, vy, BD_nx_vy, BD_nz_vy,
        dev_vz, vz, BD_nx_vz, BD_nz_vz,
        dev_tpp, tpp, BD_nx_tpp, BD_nz_tpp,
        dev_lambda, lambda,
        TotalStreamNum, AssignedStreamNum, stream,
        vx_SS_start, vy_SS_start, vz_SS_start,
        tpp_SS_start, lambda_SS_start,
        vx_SS_offset, vy_SS_offset, vz_SS_offset,
        tpp_SS_offset, lambda_SS_offset);

      // Kernekl Execution for stress
      acKernelExecforStress(
        dev_vx, vx, BD_nx_vx, BD_nz_vx,
        dev_vy, vy, BD_nx_vy, BD_nz_vy,
        dev_vz, vz, BD_nx_vz, BD_nz_vz,
        dev_tpp, tpp, BD_nx_tpp, BD_nz_tpp,
        dev_lambda, lambda,
        dx, dy, dz, dt,
        blocks, threads,
        TotalStreamNum, AssignedStreamNum, stream,
        tpp_SS_offset);

      // Copy data from device meory to host memory
      acMemcpyDToHforStress(
        dev_tpp, tpp, BD_nx_tpp, BD_nz_tpp,
        TotalStreamNum, AssignedStreamNum, stream,
        tpp_SS_start, tpp_SS_offset);

      // finish computing stress before going to the next time step
      hipDeviceSynchronize();

      // output snapshot
      // if(t%intvl==0){
      //   memcpy(snp_ptr+nsnp*BD_nx_vz*BD_ny_vz*BD_nz_vz,vz,BD_nx_vz*BD_ny_vz*BD_nz_vz*sizeof(double));
      //   nsnp++;
      // }
    }

    acHostUnRegister(vx, vy, vz, tpp, rho, lambda);
    acDeviceFree(dev_vx, dev_vy, dev_vz, dev_tpp,
      dev_lambda, dev_rho, AssignedStreamNum, stream);
    // munmap_snapshot(snp_ptr, snp_path);
  }
}
