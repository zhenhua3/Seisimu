extern "C"{
  #include<stdio.h>
  #include<unistd.h>
  #include<sys/stat.h>
  #include<sys/mman.h>
  #include<fcntl.h>
  #include<string.h>
  #include<stdlib.h>
  #include"mmap_snapshot.c"
  #include"HostRegister.cu"
  #include"GetDevicePointer.cu"
  #include"ackernel.cu"

  //double *fdc/
  //double *intvl: dz, dx, dt
  //double *modelsize : BDnDZ, BDnHX, nDZ, nHX
  void ac3d_cuda(double *vx, int BD_nx_vx, int BD_ny_vx, int BD_nz_vx,
  double *pvxbtpp,
  double *vy, int BD_nx_vy, int BD_ny_vy, int BD_nz_vy,
  double *pvybtpp,
  double *vz, int BD_nx_vz, int BD_ny_vz, int BD_nz_vz,
  double *pvzbtpp,
  double *tpp, double *ptppbvx, double *ptppbvy, double *ptppbvz,
  int BD_nx_tpp, int BD_ny_tpp, int BD_nz_tpp,
  double *rho, double *lambda, double *fdc,
  int nT, double dt, double dx, double dy, double dz, int ext,
  double *bhalf, double *ahalf, double *bfull, double *afull,
  char *snp_path, long long dim, int intvl,
  long long *threadim, long long *blockdim)
  {

    int t,nsnp=0;

    double *snp_ptr = mmap_snapshot(snp_path, dim);

    // enable zero copy access at device end
    hipError_t err = hipSetDeviceFlags(hipDeviceMapHost);
    if(err!= hipSuccess){printf("%s\n",hipGetErrorString(err));}

    // copy finite difference coefficient to constant memory
    err = hipMemcpyToSymbol(HIP_SYMBOL(dev_fdc),fdc,4*sizeof(double));
    if(err!= hipSuccess){printf("%s\n",hipGetErrorString(err));}

    // number of threads and blocks used
    dim3 threads(*threadim,*(threadim+1),*(threadim+2));
    dim3 blocks(*blockdim,*(blockdim+1),*(blockdim+2));

    // pinned host memory for faster transfer between host and device mem
    acHostRegister(vx, BD_nx_vx, BD_ny_vx, BD_nz_vx,
    vy, BD_nx_vy, BD_ny_vy, BD_nz_vy,
    vz, BD_nx_vz, BD_ny_vz, BD_nz_vz,
    tpp, BD_nx_tpp, BD_ny_tpp, BD_nz_tpp,
    rho, lambda);

    // device
    double *dev_vx, *dev_vy, *dev_vz;
    double *dev_tpp;
    double *dev_rho, *dev_lambda;

    //get device pointer
    // acGetDevicePointer(
    //   vx, vy, vz, tpp, rho, lambda,
    //   dev_vx, dev_vy, dev_vz, dev_tpp, dev_rho, dev_lambda);
      err = hipHostGetDevicePointer((void **)&dev_vx, (void *)vx,0);
      if(err!= hipSuccess){printf("%s\n",hipGetErrorString(err));}
      err = hipHostGetDevicePointer((void **)&dev_vy, (void *)vy,0);
      if(err!= hipSuccess){printf("%s\n",hipGetErrorString(err));}
      err = hipHostGetDevicePointer((void **)&dev_vz, (void *)vz,0);
      if(err!= hipSuccess){printf("%s\n",hipGetErrorString(err));}
      err = hipHostGetDevicePointer((void **)&dev_tpp, (void *)tpp,0);
      if(err!= hipSuccess){printf("%s\n",hipGetErrorString(err));}
      err = hipHostGetDevicePointer((void **)&dev_rho, (void *)rho,0);
      if(err!= hipSuccess){printf("%s\n",hipGetErrorString(err));}
      err = hipHostGetDevicePointer((void **)&dev_lambda, (void *)lambda,0);
      if(err!= hipSuccess){printf("%s\n",hipGetErrorString(err));}
    //************ time iteration ************//
    for(t=0;t<nT;t++)
    {

      // particle velocities
      ackernel_v<<<blocks,threads>>>
      (dev_vx, BD_nx_vx, BD_ny_vx, BD_nz_vx,
        dev_vy, BD_nx_vy, BD_ny_vy, BD_nz_vy,
        dev_vz, BD_nx_vz, BD_ny_vz, BD_nz_vz,
        dev_tpp, BD_nx_tpp, BD_ny_tpp, BD_nz_tpp,
        dev_rho, dx, dy, dz, dt);

      hipDeviceSynchronize();

      // stress
      ackernel_tau<<<blocks,threads>>>
      (dev_vx, BD_nx_vx, BD_ny_vx, BD_nz_vx,
        dev_vy, BD_nx_vy, BD_ny_vy, BD_nz_vy,
        dev_vz, BD_nx_vz, BD_ny_vz, BD_nz_vz,
        dev_tpp, BD_nx_tpp, BD_ny_tpp, BD_nz_tpp,
        dev_lambda, dx, dy, dz, dt);

      hipDeviceSynchronize();

      if(t%intvl==0){
        memcpy(snp_ptr+nsnp*BD_nx_vz*BD_ny_vz*BD_nz_vz,vz,BD_nx_vz*BD_ny_vz*BD_nz_vz*sizeof(double));
        nsnp++;
      }
    }

    acHostUnRegister(vx, vy, vz, tpp, rho, lambda);
    err = hipDeviceReset();
    if(err!= hipSuccess){printf("%s\n",hipGetErrorString(err));}
    munmap_snapshot(snp_ptr, snp_path);
  }
}
