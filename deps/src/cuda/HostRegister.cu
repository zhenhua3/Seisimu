
#include <hip/hip_runtime.h>
void acHostRegister(double *vx, int BD_nx_vx, int BD_ny_vx, int BD_nz_vx,
double *vy, int BD_nx_vy, int BD_ny_vy, int BD_nz_vy,
double *vz, int BD_nx_vz, int BD_ny_vz, int BD_nz_vz,
double *tpp, int BD_nx_tpp, int BD_ny_tpp, int BD_nz_tpp,
double *rho, double *lambda)
{
  int err = hipHostRegister(vx, BD_nx_vx*BD_ny_vx*BD_nz_vx*sizeof(double),hipHostRegisterMapped);
  if(err!= hipSuccess){printf("%s\n","error 3");}
  err = hipHostRegister(vy, BD_nx_vy*BD_ny_vy*BD_nz_vy*sizeof(double),hipHostRegisterMapped);
  if(err!= hipSuccess){printf("%s\n","error 4");}
  err = hipHostRegister(vz, BD_nx_vz*BD_ny_vz*BD_nz_vz*sizeof(double),hipHostRegisterMapped);
  if(err!= hipSuccess){printf("%s\n","error 5");}
  err = hipHostRegister(tpp, BD_nx_tpp*BD_ny_tpp*BD_nz_tpp*sizeof(double),hipHostRegisterMapped);
  if(err!= hipSuccess){printf("%s\n","error 6");}
  err = hipHostRegister(rho, BD_nx_tpp*BD_ny_tpp*BD_nz_tpp*sizeof(double),hipHostRegisterMapped);
  if(err!= hipSuccess){printf("%s\n","error 7");}
  err = hipHostRegister(lambda, BD_nx_tpp*BD_ny_tpp*BD_nz_tpp*sizeof(double),hipHostRegisterMapped);
  if(err!= hipSuccess){printf("%s\n","error 8");}
}

void acHostUnRegister(double *vx, double *vy, double *vz, double *tpp, double *rho, double *lambda)
{
  hipHostUnregister(vx);
  hipHostUnregister(vy);
  hipHostUnregister(vz);
  hipHostUnregister(tpp);
  hipHostUnregister(rho);
  hipHostUnregister(lambda);
}

// void elHostRegister(double *vx, int BD_nx_vx, int BD_ny_vx, int BD_nz_vx,
// double *vy, int BD_nx_vy, int BD_ny_vy, int BD_nz_vy,
// double *vz, int BD_nx_vz, int BD_ny_vz, int BD_nz_vz,
// double *tpp, int BD_nx_tpp, int BD_ny_tpp, int BD_nz_tpp,
// double *rho, double *lambda)
// {
//   // cudaHostRegister(vx, BD_nx_vx*BD_ny_vx*BD_nz_vx*sizeof(double),cudaHostRegisterMapped);
//   // cudaHostRegister(vy, BD_nx_vy*BD_ny_vy*BD_nz_vy*sizeof(double),cudaHostRegisterMapped);
//   // cudaHostRegister(vz, BD_nx_vz*BD_ny_vz*BD_nz_vz*sizeof(double),cudaHostRegisterMapped);
//   // cudaHostRegister(tpp, BD_nx_tpp*BD_ny_tpp*BD_nz_tpp*sizeof(double),cudaHostRegisterMapped);
//   // cudaHostRegister(rho, BD_nx_tpp*BD_ny_tpp*BD_nz_tpp*sizeof(double),cudaHostRegisterMapped);
//   // cudaHostRegister(lambda, BD_nx_tpp*BD_ny_tpp*BD_nz_tpp*sizeof(double),cudaHostRegisterMapped);
// }
